#include "hip/hip_runtime.h"
/////////////////////////////////////////////////////////////////////
//
//  MC Extreme  - GPU accelerated Monte-Carlo Simulation
//  
//  Author: Qianqian Fang <fangq at nmr.mgh.harvard.edu>
//  History: 
//    2009/02/14 initial version written in BrookGPU
//    2009/02/15 translated to CUDA
//    2009/02/20 translated to Brook+
//    2009/02/21 added MT random number generator initial version
//    2009/02/24 MT rand now works fine, added FAST_MATH
//    2009/02/25 added CACHE_MEDIA read
//    2009/02/27 early support of boundary reflection
//    2009/03/02 added logistic-map based random number generator
//
// License: unpublished version, use by author's permission only
//
/////////////////////////////////////////////////////////////////////

#include <stdio.h>
#include "br2cu.h"
#include "tictoc.h"

#ifdef USE_MT_RAND
#include "mt_rand_s.cu"
#else
#include "logistic_rand.cu"
#endif

// dimension of the target domain
#define DIMX 128
#define DIMY 128
#define DIMZ 128
/*
#define DIMX 256
#define DIMY 256
#define DIMZ 256
*/

#define DIMYZ (DIMY*DIMZ)
#define DIMXYZ (DIMX*DIMY*DIMZ)
#define INDXYZ(ii,jj,kk)  ((ii)*DIMYZ+(jj)*DIMZ+(kk))
#define MAX_MT_RAND 4294967296
#define R_MAX_MT_RAND 2.32830643653870e-10
#define TWO_PI 6.28318530717959f
#define EPS    (1e-10f)

#ifdef __DEVICE_EMULATION__
#define MAX_THREAD 1
#else
#define MAX_THREAD 128
//#define MAX_THREAD 256
#endif
#define MAX_EVENT  1
#define MAX_PROP   256


#ifdef CACHE_MEDIA
//#define MAX_MEDIA_CACHE   61440  /*52k for local media read*/
#define MAX_MEDIA_CACHE   40000  /*52k for local media read*/
#define MAX_WRITE_CACHE   (MAX_MEDIA_CACHE>>4)
#define MEDIA_BITS  8            /*theoretically one can use smaller bits to pack more media*/
#define MEDIA_PACK  ((8/MEDIA_BITS)>>1)            /*one byte packs 2^MEDIA_PACK voxel*/
#define MEDIA_MOD   ((1<<MEDIA_PACK)-1)    /*one byte packs 2^MEDIA_PACK voxel*/
#define MEDIA_MASK  ((1<<(MEDIA_BITS))-1)
#endif

#define MINUS_SAME_VOXEL -9999.f

#define GPUDIV(a,b)     __fdividef((a),(b))

#ifdef  FAST_MATH      /*define this to use the fast math functions*/
#define	GPULOG(x)       __logf(x)
#define GPUSIN(x)       __sinf(x)
#define	GPUSINCOS(x,a,b)       __sincosf(x,a,b)
#else
#define GPULOG(x)       logf(x)
#define GPUSIN(x)       sinf(x)
#define GPUSINCOS(x,a,b)      sincosf(x,a,b)
#endif


typedef unsigned char uchar;

/******************************
typedef struct PhotonData {
  float4 pos;  // x,y,z,weight
  float4 dir;  // ix,iy,iz,dummy
  float3 len; // resid,tot,count
  uint   seed; // random seed
} Photon;
******************************/

__constant__ float3 gproperty[MAX_PROP];

#ifdef CACHE_MEDIA
__constant__ uchar  gmediacache[MAX_MEDIA_CACHE];
#endif

// pass as many pre-computed values as possible to utilize the constant memory 

kernel void mcx_main_loop(int totalmove,uchar media[],float field[],float3 vsize,float minstep, 
     float lmax, float gg, float gg2,float ggx2, float one_add_gg2, float one_sub_gg2, float one_sub_gg,
     float4 p0,float4 c0,float3 maxidx,uint3 cp0,uint3 cp1,uint2 cachebox,uchar doreflect,
     uint n_seed[],float4 n_pos[],float4 n_dir[],float3 n_len[]){

     int idx= blockDim.x * blockIdx.x + threadIdx.x;

     float4 npos=n_pos[idx];
     float4 ndir=n_dir[idx];
     float3 nlen=n_len[idx];
     float4 npos0;
     float3 htime;

     int i, idx1d, idx1dold,idxorig, mediaid;

     float flipdir,n1,Rtotal;
#ifdef CACHE_MEDIA
     int incache=0,incache0=0,cachebyte=-1,cachebyte0=-1;
#endif

#ifdef USE_MT_RAND
     uint   ran;
#else
     uint  randid=0;
     RandType ran, t[RAND_BUF_LEN],tnew[RAND_BUF_LEN];
#endif

     float3 prop;

     float len,cphi,sphi,theta,stheta,ctheta,tmp0,tmp1;

#ifdef USE_MT_RAND
     mt19937si(n_seed[idx]);
     __syncthreads();
#else
     logistic_init(t,tnew,n_seed,idx);
#endif

     // assuming the initial positions are within the domain
     idx1d=int(floorf(npos.x)*DIMYZ+floorf(npos.y)*DIMZ+floorf(npos.z));
     idxorig=idx1d;
     mediaid=media[idx1d];

#ifdef CACHE_MEDIA
     if(npos.x>=cp0.x && npos.x<=cp1.x && npos.y>=cp0.y && npos.y<=cp1.y && npos.z>=cp0.z && npos.z<=cp1.z){
	  incache=1;
          incache0=1;
          cachebyte=int(floorf(npos.x-cp0.x)*cachebox.y+floorf(npos.y-cp0.y)*cachebox.x+floorf(npos.z-cp0.z));
          cachebyte0=cachebyte;
          mediaid=gmediacache[cachebyte];
     }
#endif

     if(mediaid==0) {
          return; /* the initial position is not within the medium*/
     }

     // using "while(nlen.z<totalmove)" loop will make this 4 times slower with the same amount of photons

     for(i=0;i<totalmove;i++){
	  if(nlen.x<=0.f) {  /* if this photon has finished the current jump */

#ifdef USE_MT_RAND
	       ran=mt19937s(); /*random number [0,MAX_MT_RAND)*/
   	       nlen.x=-GPULOG(ran*R_MAX_MT_RAND); /*probability of the next jump*/
#else
               logistic_rand(t,tnew,RAND_BUF_LEN-1); /*create 3 random numbers*/
               randid=0;

               ran=logistic_uniform(t[0]);                             /*order 2,0,1, small shuffle, not really help*/
               nlen.x= ((ran==0.f)?(-GPULOG(t[0])):(-GPULOG(ran)));
#endif

#ifdef __DEVICE_EMULATION__
               printf("1 %20.16e \n",nlen.x);
#endif


	       if(npos.w<1.f){ /*weight*/
                       /*random arimuthal angle*/
#ifdef USE_MT_RAND
                       ran=mt19937s();
		       tmp0=TWO_PI*ran*R_MAX_MT_RAND; /*will be reused to minimize register*/
#else
                       ran=t[2]; /*random number [0,MAX_MT_RAND)*/
                       tmp0=TWO_PI*logistic_uniform(ran); /*will be reused to minimize register*/
#endif

#ifdef __DEVICE_EMULATION__
               printf("2 %20.16e\n",tmp0);
#endif
                       GPUSINCOS(tmp0,&sphi,&cphi);

                       /*Henyey-Greenstein Phase Function, "Handbook of Optical Biomedical Diagnostics",2002,Chap3,p234*/
                       /*see Boas2002*/

#ifdef USE_MT_RAND
		       ran=mt19937s();
#else
                       ran=t[4]; /*random number [0,MAX_MT_RAND)*/
#endif

                       if(gg>EPS){
#ifdef USE_MT_RAND
		           tmp0=GPUDIV(one_sub_gg2,(one_sub_gg+ggx2*ran*R_MAX_MT_RAND));
#else
                           tmp0=GPUDIV(one_sub_gg2,(one_sub_gg+ggx2*logistic_uniform(ran) ));
#endif

#ifdef __DEVICE_EMULATION__
               printf("3 %20.16e\n",tmp0);
#endif

		           tmp0*=tmp0;
		           tmp0=GPUDIV((one_add_gg2-tmp0),ggx2);
		           theta=acosf(tmp0);
		           stheta=GPUSIN(theta);
		           ctheta=tmp0;
                       }else{
#ifdef USE_MT_RAND
			   theta=TWO_PI*ran*R_MAX_MT_RAND;
#else
                           theta=TWO_PI*logistic_uniform(ran);
#endif

                           GPUSINCOS(theta,&stheta,&ctheta);
                       }
		       if( ndir.z>-1.f+EPS && ndir.z<1.f-EPS ) {
		           tmp0=1.f-ndir.z*ndir.z;   /*reuse tmp to minimize registers*/
		           tmp1=rsqrtf(tmp0);
		           tmp1=stheta*tmp1;
//			   if(stheta>1e-20) {  /*strange: if stheta=0, I will get nan :(  FQ */
			     ndir=float4(
				tmp1*(ndir.x*ndir.z*cphi - ndir.y*sphi) + ndir.x*ctheta,
				tmp1*(ndir.y*ndir.z*cphi + ndir.x*sphi) + ndir.y*ctheta,
				-tmp1*tmp0*cphi                         + ndir.z*ctheta,
				ndir.w
				);
//                             }
		       }else{
			   ndir=float4(stheta*cphi,stheta*sphi,ctheta,ndir.w);
 		       }
                       ndir.w++;
	       }
	  }


          n1=prop.z;
	  prop=gproperty[mediaid];
	  len=minstep*prop.y;

          npos0=npos;
	  if(len>nlen.x){  /*scattering ends in this voxel*/
               tmp0=GPUDIV(nlen.x,prop.y);
   	       npos=float4(npos.x+ndir.x*tmp0,npos.y+ndir.y*tmp0,npos.z+ndir.z*tmp0,npos.w*expf(-prop.x * tmp0 ));
	       nlen.x=MINUS_SAME_VOXEL;
	       nlen.y+=tmp0;
	  }else{                      /*otherwise, move minstep*/
   	       npos=float4(npos.x+ndir.x,npos.y+ndir.y,npos.z+ndir.z,npos.w*expf(-prop.x * minstep ));
	       nlen.x-=len;     /*remaining probability*/
	       nlen.y+=minstep; /*total moved length along the current jump*/
               idx1dold=idx1d;
               idx1d=int(floorf(npos.x)*DIMYZ+floorf(npos.y)*DIMZ+floorf(npos.z));
#ifdef CACHE_MEDIA  
               if(npos.x>=cp0.x && npos.x<=cp1.x && npos.y>=cp0.y && npos.y<=cp1.y && npos.z>=cp0.z && npos.z<=cp1.z){
                    incache=1;
                    cachebyte=int(floorf(npos.x-cp0.x)*cachebox.y+floorf(npos.y-cp0.y)*cachebox.x+floorf(npos.z-cp0.z));
               }else{
		    incache=0;
               }
#endif
	  }

#ifdef CACHE_MEDIA
          mediaid=incache?gmediacache[cachebyte]:media[idx1d];
#else
          mediaid=media[idx1d];
#endif

	  if(mediaid==0||nlen.y>lmax||npos.x<0||npos.y<0||npos.z<0||npos.x>maxidx.x||npos.y>maxidx.y||npos.z>maxidx.z){
	      /*if hit the boundary or exit the domain, launch a new one*/

              /*time to hit the wall in each direction*/
              htime.x=(ndir.x>EPS||ndir.x<-EPS)?(floorf(npos.x)+(ndir.x>0.f)-npos.x)/ndir.x:1e10; /*this approximates*/
              htime.y=(ndir.y>EPS||ndir.y<-EPS)?(floorf(npos.y)+(ndir.y>0.f)-npos.y)/ndir.y:1e10f;
              htime.z=(ndir.z>EPS||ndir.z<-EPS)?(floorf(npos.z)+(ndir.z>0.f)-npos.z)/ndir.z:1e10f;
              tmp0=fminf(fminf(htime.x,htime.y),htime.z);
              flipdir=(tmp0==htime.x?1.f:(tmp0==htime.y?2.f:(tmp0==htime.z&&idx1d!=idx1dold)?3.f:0.f));
              prop=gproperty[mediaid];

#ifdef __DEVICE_EMULATION__
//              printf("--> ID%d J%d C%d len %f flip %d %f!=%f dir=%f %f %f \n",idx,(int)ndir.w,
//                  (int)nlen.z,nlen.y, (int)flipdir, n1,prop.z,ndir.x,ndir.y,ndir.z);
#endif

              /*I don't have the luxury to declare more vars in a kernel, so, I recycled some of old ones*/

              if(doreflect&&nlen.y<lmax && flipdir>0.f && n1!=prop.z){
                  tmp0=n1*n1;
                  tmp1=prop.z*prop.z;
                  if(flipdir>=3.f) { /*flip in z axis*/
                     cphi=fabs(ndir.z);
                     sphi=ndir.x*ndir.x+ndir.y*ndir.y;
                     ndir.z=-ndir.z;
                  }else if(flipdir>=2.f){ /*flip in y axis*/
                     cphi=fabs(ndir.y);
       	       	     sphi=ndir.x*ndir.x+ndir.z*ndir.z;
                     ndir.y=-ndir.y;
                  }else if(flipdir>=1.f){ /*flip in x axis*/
                     cphi=fabs(ndir.x);               /*cos(si)*/
                     sphi=ndir.y*ndir.y+ndir.z*ndir.z; /*sin(si)^2*/
                     ndir.x=-ndir.x;
                  }
                  npos=npos0;   /*move back*/
                  idx1d=idx1dold;
                  len=1.f-GPUDIV(tmp0,tmp1)*sphi;   /*1-[n1/n2*sin(si)]^2*/
                  if(len>0.f) {
                     ctheta=tmp0*cphi*cphi+tmp1*len;
                     stheta=2.f*n1*prop.z*cphi*sqrtf(len);
                     Rtotal=GPUDIV(ctheta-stheta,ctheta+stheta);
       	       	     ctheta=tmp1*cphi*cphi+tmp0*len;
       	       	     Rtotal=(Rtotal+GPUDIV(ctheta-stheta,ctheta+stheta))/2.f;
#ifdef __DEVICE_EMULATION__
//printf("  dir=%f %f %f htime=%f %f %f Rs=%f\n",ndir.x,ndir.y,ndir.z,htime.x,htime.y,htime.z,Rtotal);
//printf("  ID%d J%d C%d flip=%3f (%d %d) cphi=%f sphi=%f npos=%f %f %f npos0=%f %f %f\n",idx,(int)ndir.w,(int)nlen.z,
//            flipdir,idx1dold,idx1d,cphi,sphi,npos.x,npos.y,npos.z,npos0.x,npos0.y,npos0.z);
#endif
                     npos.w*=Rtotal;
                  } /* else, total internal reflection, no loss*/
                  mediaid=media[idx1d];
                  prop=gproperty[mediaid];
                  n1=prop.z;
              }else{
	          npos=p0;
	          ndir=c0;
	          nlen=float3(0.f,0.f,nlen.z+1);
                  idx1d=idxorig;
#ifdef CACHE_MEDIA
	          cachebyte=cachebyte0;
	          incache=incache0;
#endif
              }
	  }else if(nlen.x>0){
#ifdef __DEVICE_EMULATION__
//    printf("field add to %d->%f(%d)\n",idx1d,npos.w,(int)nlen.z);
#endif
              field[idx1d]+=npos.w;
	  }
     }
#ifdef USE_MT_RAND
     n_seed[idx]=(ran&0xffffffffu);
#else
     n_seed[idx]=ran*0xffffffffu;
#endif
     n_pos[idx]=npos;
     n_dir[idx]=ndir;
     n_len[idx]=nlen;
}

void savedata(float *dat,int len,char *name){
     FILE *fp;
     fp=fopen(name,"wb");
     fwrite(dat,sizeof(float),len,fp);
     fclose(fp);
}

void mcx_error(int id,char *msg){
     printf("MCX ERROR(%d):%s\n",id,msg);
     exit(id);
}

int main (int argc, char *argv[]) {

     int    nthread=1024;   /*the default total thread number*/
     uint3  griddim;        /*the grid size*/

     float3 vsize=float3(1.f,1.f,1.f);
     float  minstep=1.f;
     float  lmax=1000.f;
     float  gg=0.98f;
     float4 p0=float4(DIMX/2,DIMY/2,DIMZ/4,1.f);
     float4 c0=float4(0.f,0.f,1.f,0.f);
     float3 maxidx=float3(DIMX-1,DIMY-1,DIMZ-1);
     float3 property[MAX_PROP]={float3(0.f,0.f,1.0f),float3(0.009f,0.75f,1.37f),  // the 1st is air
                                float3(0.006f,0.75f,1.37f),float3(0.009f,0.95f,1.37f)};

     int i,j,k;
     int totalmove=MAX_EVENT;
     int photoncount=0;
     int tic;
     uint3 cp0=uint3(DIMX/2-3,DIMY/2-3,DIMZ/4),cp1=uint3(DIMX/2+4,DIMY/2+4,DIMZ/4+5);
//     uint3 cp0=uint3(DIMX/2-35,DIMY/2-35,DIMZ/4-1),cp1=uint3(DIMX/2+35,DIMY/2+35,DIMZ/4+65);
     uint2 cachebox;

     dim3 GridDim;
     dim3 BlockDim;

     uchar  media[DIMXYZ];
     float  field[DIMXYZ];
#ifdef CACHE_MEDIA
     int count;
     uchar  mediacache[MAX_MEDIA_CACHE];
#endif

     float4 *Ppos;
     float4 *Pdir;
     float3 *Plen;
     uint   *Pseed;

     if(argc>1){
	   totalmove=atoi(argv[1]); //number of the total move per thread, this is not the photon number
           if(argc>2){
               nthread=atoi(argv[2]);
           }
     }
     if(nthread<=0) {
           mcx_error(1,"total thread number must be positive (recommended to be multiple of 32, say 1024)");
     }
     GridDim.x=nthread/MAX_THREAD;
     BlockDim.x=MAX_THREAD;

     Ppos=(float4*)malloc(sizeof(float4)*nthread);
     Pdir=(float4*)malloc(sizeof(float4)*nthread);
     Plen=(float3*)malloc(sizeof(float3)*nthread);
     Pseed=(uint*)malloc(sizeof(uint)*nthread*RAND_BUF_LEN);

#ifdef CACHE_MEDIA
     printf("requested constant memory cache: %d (max allowed %d)\n",
         (cp1.x-cp0.x+1)*(cp1.y-cp0.y+1)*(cp1.z-cp0.z+1),(MAX_MEDIA_CACHE<<MEDIA_PACK));

     if((cp1.x-cp0.x+1)*(cp1.y-cp0.y+1)*(cp1.z-cp0.z+1)> (MAX_MEDIA_CACHE<<MEDIA_PACK)){
	printf("the requested cache size is too big\n");
	exit(1);
     }
#endif

     uchar *gmedia;
     hipMalloc((void **) &gmedia, sizeof(uchar)*(DIMXYZ));
     float *gfield;
     hipMalloc((void **) &gfield, sizeof(float)*(DIMXYZ));

     float4 *gPpos;
     hipMalloc((void **) &gPpos, sizeof(float4)*nthread);
     float4 *gPdir;
     hipMalloc((void **) &gPdir, sizeof(float4)*nthread);
     float3 *gPlen;
     hipMalloc((void **) &gPlen, sizeof(float3)*nthread);
     uint   *gPseed;
     hipMalloc((void **) &gPseed, sizeof(uint)*nthread*RAND_BUF_LEN);


     memset(field,0,sizeof(float)*DIMXYZ);
     memset(media,0,sizeof(uchar)*DIMXYZ);

     for (i=DIMX/2-25; i<DIMX/2+25; i++)
      for (j=DIMX/2-25; j<DIMX/2+25; j++)
       for (k=DIMZ/4; k<DIMZ/4+50; k++) {
           media[INDXYZ(i,j,k)]=1; 
       }

     cachebox.x=(cp1.z-cp0.z+1);
     cachebox.y=(cp1.y-cp0.y+1)*(cp1.z-cp0.z+1);

#ifdef CACHE_MEDIA
     count=0;
     memset(mediacache,0,MAX_MEDIA_CACHE);

     /*only use 1-byte to store media info, unpacking bits on-the-fly turned out to be expensive in gpu*/

     for (i=cp0.x; i<=cp1.x; i++)
      for (j=cp0.y; j<=cp1.y; j++)
       for (k=cp0.z; k<=cp1.z; k++) {
//         printf("[%d %d %d]: %d %d %d %d (%d)\n",i,j,k,count,MEDIA_MASK,count>>MEDIA_PACK,(count & MEDIA_MOD)*MEDIA_BITS,
//                (media[INDXYZ(i,j,k)] & MEDIA_MASK )<<((count & MEDIA_MOD)*MEDIA_BITS) );
           mediacache[count>>MEDIA_PACK] |=  (media[INDXYZ(i,j,k)] & MEDIA_MASK )<<((count & MEDIA_MOD)*MEDIA_BITS );
           count++;
       }
#endif

     srand(time(0));
     for (i=0; i<nthread; i++) {
	   Ppos[i]=p0;  /* initial position */
           Pdir[i]=c0;
           Plen[i]=float3(0.f,0.f,0.f);
     }
     for (i=0; i<nthread*RAND_BUF_LEN; i++) {
	   Pseed[i]=rand();
     }
     tic=GetTimeMillis();

     hipMemcpy(gPpos,  Ppos,  sizeof(float4)*nthread,  hipMemcpyHostToDevice);
     hipMemcpy(gPdir,  Pdir,  sizeof(float4)*nthread,  hipMemcpyHostToDevice);
     hipMemcpy(gPlen,  Plen,  sizeof(float3)*nthread,  hipMemcpyHostToDevice);
     hipMemcpy(gPseed, Pseed, sizeof(uint)*nthread*RAND_BUF_LEN,  hipMemcpyHostToDevice);
     hipMemcpy(gfield, field, sizeof(float)*DIMXYZ, hipMemcpyHostToDevice);
     hipMemcpy(gmedia, media, sizeof(uchar)*DIMXYZ,hipMemcpyHostToDevice);
     hipMemcpyToSymbol(HIP_SYMBOL(gproperty), property, MAX_PROP*sizeof(float3), 0, hipMemcpyHostToDevice);
#ifdef CACHE_MEDIA
     hipMemcpyToSymbol(HIP_SYMBOL(gmediacache), mediacache, MAX_MEDIA_CACHE, 0, hipMemcpyHostToDevice);
#endif

     printf("complete hipMemcpy : %d ms\n",GetTimeMillis()-tic);

     mcx_main_loop<<<GridDim,BlockDim>>>(totalmove,gmedia,gfield,vsize,minstep,lmax,gg,gg*gg,2.f*gg,\
        	 1.f+gg*gg,1.f-gg*gg,1.f-gg,p0,c0,maxidx,cp0,cp1,cachebox,0,gPseed,gPpos,gPdir,gPlen);

     printf("complete launching kernels : %d ms\n",GetTimeMillis()-tic);

     hipMemcpy(Ppos,  gPpos, sizeof(float4)*nthread, hipMemcpyDeviceToHost);

     printf("complete retrieving pos : %d ms\n",GetTimeMillis()-tic);

     hipMemcpy(Pdir,  gPdir, sizeof(float4)*nthread, hipMemcpyDeviceToHost);
     hipMemcpy(Plen,  gPlen, sizeof(float3)*nthread, hipMemcpyDeviceToHost);
     hipMemcpy(Pseed, gPseed,sizeof(uint)*nthread*RAND_BUF_LEN,   hipMemcpyDeviceToHost);
     hipMemcpy(field, gfield,sizeof(float)*DIMXYZ,hipMemcpyDeviceToHost);

     printf("complete retrieving all : %d ms\n",GetTimeMillis()-tic);

     for (i=0; i<nthread; i++) {
	  photoncount+=(int)Plen[i].z;
     }
     totalmove=nthread<16?nthread:16;
     for (i=0; i<totalmove; i++) {
           printf("% 4d[A% f % f % f]C%3d J%3d% 8f(P% 6.3f % 6.3f % 6.3f)T% 5.3f L% 5.3f %f %f\n", i,
            Pdir[i].x,Pdir[i].y,Pdir[i].z,(int)Plen[i].z,(int)Pdir[i].w,Ppos[i].w, 
            Ppos[i].x,Ppos[i].y,Ppos[i].z,Plen[i].y,Plen[i].x,(float)Pseed[i], Pdir[i].x*Pdir[i].x+Pdir[i].y*Pdir[i].y+Pdir[i].z*Pdir[i].z);
     }
     printf("simulated %d photons\n",photoncount);
     savedata(field,DIMX*DIMY*DIMZ,"field.dat");

     hipFree(gmedia);
#ifdef CACHE_MEDIA
     hipFree(gmediacache);
#endif
     hipFree(gfield);
     hipFree(gPpos);
     hipFree(gPdir);
     hipFree(gPlen);
     hipFree(gPseed);
     free(Ppos);
     free(Pdir);
     free(Plen);
     free(Pseed);

     return 0;
}
