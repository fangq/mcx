/////////////////////////////////////////////////////////////////////
//
//  Monte-Carlo Extreme (MCX) - a GPU accelerated Monte-Carlo Simulation
//  Random Number Generator Benchmark
//
//  Author: Qianqian Fang <fangq at nmr.mgh.harvard.edu>
//  History: 
//     2009/04/16 test for the speed of the two RNG in MCX
//
/////////////////////////////////////////////////////////////////////

#include <stdio.h>
#include "br2cu.h"

#define USE_OS_TIMER  /* use MT19937 RNG */
#include "tictoc.c"

#define RAND_TEST_LEN 5

#ifdef USE_MT_RAND  /* use MT19937 RNG */

//===================================================================
// GPU kernels for MT19937
//===================================================================

#include "mt_rand_s.cu"

kernel void bench_rng(uint seed[],float output[],int loop){
     int idx= blockDim.x * blockIdx.x + threadIdx.x;
     int base=idx*loop;
     uint ran;
     float res;
     int i;

     mt19937si(seed,idx);
     for(i=0;i<loop;i++){
          ran=mt19937s();
	  res=ran*R_MAX_MT_RAND;
#ifdef GLOBAL_WRITE
	  output[base+i]=res;
    #ifdef __DEVICE_EMULATION__
          printf("%d = %d %f\n",i,base+i,output[base+i]);
    #endif
#endif
     }
#ifndef GLOBAL_WRITE  /*to prevent the compiler from optimizing the rand*/
     output[base]=  res;
#endif     
}

#else   /* use Logistic-map lattice RNG */

//===================================================================
// GPU kernels for Logistic-map lattice LL3 and LL5
//===================================================================
//#include "logistic_rand_ring3.cu" // for LL3 RNG

#include "logistic_rand.cu"

kernel void bench_rng(uint seed[],float output[],int loop){
     int idx= blockDim.x * blockIdx.x + threadIdx.x;
     int base=idx*loop;
     int i;
     float c=0.f;
     
     RandType t[RAND_TEST_LEN],tnew[RAND_TEST_LEN];
     RandType ran[RAND_TEST_LEN];
     logistic_init(t,tnew,seed,idx);

     for(i=0;i<loop;i+=5){
          logistic_step(t,tnew,RAND_TEST_LEN-1); /*create 3 random numbers*/
	  ran[0]=logistic_uniform(t[0]);
	  ran[1]=logistic_uniform(t[1]);
	  ran[2]=logistic_uniform(t[2]);
	  ran[3]=logistic_uniform(t[3]);
	  ran[4]=logistic_uniform(t[4]);
	  c+=ran[0];
#ifdef GLOBAL_WRITE
	  output[base+i]=  ran[0];
	  output[base+i+1]=ran[1];
	  output[base+i+2]=ran[2];
	  output[base+i+3]=ran[3];
	  output[base+i+4]=ran[4];
    #ifdef __DEVICE_EMULATION__
          printf("%d = %d %f %f %f\n",i,base+i,ran[0],ran[1],ran[2]);
    #endif
#endif
     }
#ifndef GLOBAL_WRITE  /*to prevent the compiler from optimizing the rand*/
     output[base+i]=  c;
#endif
}

#endif

//===================================================================
// utility functions
//===================================================================

void usage(char *exename){
	printf("usage: %s <num_block|128> <num_thread|128> <rand_per_thread|10000> <num_repeat|10>\n",exename);
}

void mcx_savedata(float *dat,int len,char *name){
     FILE *fp;
     fp=fopen(name,"wb");
     fwrite(dat,sizeof(float),len,fp);
     fclose(fp);
}

//===================================================================
// main program
//===================================================================

int main(int argc, char *argv[]){
    dim3 griddim=128, blockdim=128;
    int count=999,repeat=10, threadnum,tic,tic2,toc;
    uint   *Pseed;
    uint   *gPseed;
    float  *Poutput;
    float  *gPoutput;
    double totalrand;
    int i;

    // parse arguments
    
    if(argc==1){
	usage(argv[0]);
	exit(0);
    }
    if(argc>=2) griddim.x=atoi(argv[1]);
    if(argc>=3) blockdim.x=atoi(argv[2]);
    if(argc>=4) count=atoi(argv[3]);
    if(argc>=5) repeat=atoi(argv[4]);
    
    if(RAND_TEST_LEN>0)
        count=(count/RAND_TEST_LEN)*RAND_TEST_LEN; // make count modulo of 5

    threadnum=griddim.x*blockdim.x;

    // allocate CPU and GPU arrays
    
    Pseed=(uint*)malloc(sizeof(uint)*threadnum*RAND_TEST_LEN);
    hipMalloc((void **) &gPseed, sizeof(uint)*threadnum*RAND_TEST_LEN);
    Poutput=(float*)malloc(sizeof(float)*threadnum*count);
    hipMalloc((void **) &gPoutput, sizeof(float)*threadnum*count);

    // initialize seeds
        
    srand(time(0));
    for (i=0; i<threadnum*RAND_TEST_LEN; i++){
	   Pseed[i]=rand();
    }

    // copy CPU data to GPU
    
    tic=StartTimer();
    totalrand=(double)threadnum*count*repeat;
    printf("total thread=%d, total rand num=%f\n",threadnum,totalrand);
    hipMemcpy(gPseed, Pseed, sizeof(uint)*threadnum*RAND_TEST_LEN,hipMemcpyHostToDevice);

    printf("init complete : %d ms\n",GetTimeMillis()-tic);

    // begin benchmark
    
    tic2=StartTimer();
    for(i=0;i<repeat;i++)
        bench_rng<<<griddim,blockdim>>>(gPseed,gPoutput,count);

    // get only one element to make sure all kernels are complete
    
    hipMemcpy(Pseed,gPseed, sizeof(uint),hipMemcpyDeviceToHost);
    toc=GetTimeMillis()-tic2;

    // take results back to CPU
        
    printf("kernel complete: %d ms\nspeed: %f random numbers per second\n",\
        toc, (1000./toc)*totalrand);
//    hipMemcpy(Pseed, gPseed,sizeof(uint)*threadnum*RAND_TEST_LEN,hipMemcpyDeviceToHost);
#ifdef GLOBAL_WRITE
    hipMemcpy(Poutput,gPoutput, sizeof(float)*threadnum*count,hipMemcpyDeviceToHost);
#endif

#ifdef GLOBAL_WRITE
    // dump random numbers to disk
    mcx_savedata(Poutput,threadnum*count,"randomnum.bin");
    printf("dump data complete: %d ms\n",GetTimeMillis()-tic);
#endif

    // memory clean-up
    
    hipFree(gPseed);
    hipFree(gPoutput);
    free(Pseed);
    free(Poutput);
    return 0;
}
