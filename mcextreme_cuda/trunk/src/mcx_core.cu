#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
//
//  Monte Carlo eXtreme (MCX)  - GPU accelerated Monte Carlo 3D photon migration
//  Author: Qianqian Fang <fangq at nmr.mgh.harvard.edu>
//
//  Reference (Fang2009):
//        Qianqian Fang and David A. Boas, "Monte Carlo Simulation of Photon 
//        Migration in 3D Turbid Media Accelerated by Graphics Processing 
//        Units," Optics Express, vol. 17, issue 22, pp. 20178-20190 (2009)
//
//  mcx_core.cu: GPU kernels and CUDA host code
//
//  License: GNU General Public License v3, see LICENSE.txt for details
//
////////////////////////////////////////////////////////////////////////////////

#include "br2cu.h"
#include "mcx_core.h"
#include "tictoc.h"
#include "mcx_const.h"

#ifdef USE_MT_RAND
#include "mt_rand_s.cu"     // use Mersenne Twister RNG (MT)
#else
#include "logistic_rand.cu" // use Logistic Lattice ring 5 RNG (LL5)
#endif

// optical properties saved in the constant memory
// {x}:mua,{y}:mus,{z}:anisotropy (g),{w}:refractive index (n)
__constant__ float4 gproperty[MAX_PROP];

__constant__ float4 gdetpos[MAX_DETECTORS];

// kernel constant parameters
__constant__ MCXParam gcfg[1];

extern __shared__ float sharedmem[]; //max 64 tissue types when block size=64

// tested with texture memory for media, only improved 1% speed
// to keep code portable, use global memory for now
// also need to change all media[idx1d] to tex1Dfetch() below
//texture<uchar, 1, hipReadModeElementType> texmedia;

__device__ inline void atomicadd(float* address, float value){

#if __CUDA_ARCH__ >= 200 // for Fermi, atomicAdd supports floats

  atomicAdd(address,value);

#elif __CUDA_ARCH__ >= 110

// float-atomic-add from 
// http://forums.nvidia.com/index.php?showtopic=158039&view=findpost&p=991561
  float old = value;  
  while ((old = atomicExch(address, atomicExch(address, 0.0f)+old))!=0.0f);

#endif

}

__device__ inline void clearpath(float *p,int maxmediatype){
      uint i;
      for(i=0;i<maxmediatype;i++)
      	   p[i]=0.f;
}

__device__ inline void clearcache(float *p,int len){
      uint i;
      if(threadIdx.x==0)
        for(i=0;i<len;i++)
      	   p[i]=0.f;
}

#ifdef  USE_CACHEBOX
__device__ inline void savecache(float *data,float *cache){
      uint x,y,z;
      if(threadIdx.x==0){
        for(z=gcfg->cp0.z;z<=gcfg->cp1.z;z++)
           for(y=gcfg->cp0.y;y<=gcfg->cp1.y;y++)
              for(x=gcfg->cp0.x;x<=gcfg->cp1.x;x++){
                 atomicadd(data+z*gcfg->dimlen.y+y*gcfg->dimlen.x+x,
		    cache[(z-gcfg->cp0.z)*gcfg->cachebox.y+(y-gcfg->cp0.y)*gcfg->cachebox.x+(x-gcfg->cp0.x)]);
	      }
      }
}
#endif

#ifdef SAVE_DETECTORS
__device__ inline uint finddetector(MCXpos *p0){
      uint i;
      for(i=0;i<gcfg->detnum;i++){
      	if((gdetpos[i].x-p0->x)*(gdetpos[i].x-p0->x)+
	   (gdetpos[i].y-p0->y)*(gdetpos[i].y-p0->y)+
	   (gdetpos[i].z-p0->z)*(gdetpos[i].z-p0->z) < gdetpos[i].w){
	        return i+1;
	   }
      }
      return 0;
}

__device__ inline void savedetphoton(float n_det[],uint *detectedphoton,float weight,float *ppath,MCXpos *p0){
      uint j,baseaddr=0;
      j=finddetector(p0);
      if(j){
	 baseaddr=atomicAdd(detectedphoton,1);
	 if(baseaddr<gcfg->maxdetphoton){
	    baseaddr*=gcfg->maxmedia+2;
	    n_det[baseaddr++]=j;
	    n_det[baseaddr++]=weight;
	    for(j=0;j<gcfg->maxmedia;j++){
		n_det[baseaddr+j]=ppath[j]; // save partial pathlength to the memory
	    }
	 }
      }
}
#endif

__device__ inline void launchnewphoton(MCXpos *p,MCXdir *v,MCXtime *f,uint *idx1d,
           uchar *mediaid,uchar isdet, float ppath[],float energyloss[],float n_det[],uint *dpnum){

      *energyloss+=p->w;  // sum all the remaining energy
#ifdef SAVE_DETECTORS
      // let's handle detectors here
      if(gcfg->savedet){
         if(*mediaid==0 && isdet)
	      savedetphoton(n_det,dpnum,v->nscat,ppath,p);
	 clearpath(ppath,gcfg->maxmedia);
      }
#endif
      *((float4*)p)=gcfg->ps;
      *((float4*)v)=gcfg->c0;
      *((float4*)f)=float4(0.f,0.f,gcfg->minaccumtime,f->ndone+1);
      *idx1d=gcfg->idx1dorig;
      *mediaid=gcfg->mediaidorig;
}

/*
   this is the core Monte Carlo simulation kernel, please see Fig. 1 in Fang2009
*/
kernel void mcx_main_loop(int nphoton,int ophoton,uchar media[],float field[],
     float genergy[],uint n_seed[],float4 n_pos[],float4 n_dir[],float4 n_len[],
     float n_det[], uint *detectedphoton){

     int idx= blockDim.x * blockIdx.x + threadIdx.x;

     MCXpos  p,p0;//{x,y,z}: coordinates, w:packet weight
     MCXdir  v;   //{x,y,z}: unitary direction vector, nscat:total scat event
     MCXtime f;   //tscat: remaining scattering time,t: photon elapse time, 
                  //tnext: next accumulation time, ndone: completed photons
     float  energyloss=genergy[idx<<1];
     float  energyabsorbed=genergy[(idx<<1)+1];

     uint np,idx1d, idx1dold;   //idx1dold is related to reflection
     //np=nphoton+((idx==blockDim.x*gridDim.x-1) ? ophoton: 0);

#ifdef TEST_RACING
     int cc=0;
#endif
     uchar  mediaid,isdet=0;
     char   medid=-1;
     float  atten;         //can be taken out to minimize registers
     float  n1;   //reflection var

     //for MT RNG, these will be zero-length arrays and be optimized out
     RandType t[RAND_BUF_LEN],tnew[RAND_BUF_LEN];
     Medium prop;    //can become float2 if no reflection

     float len,cphi,sphi,theta,stheta,ctheta,tmp0,tmp1;

     float *ppath   =sharedmem+threadIdx.x*gcfg->maxmedia;
#ifdef  USE_CACHEBOX
     float *cachebox=sharedmem+blockDim.x*gcfg->maxmedia;
     if(gcfg->skipradius2>EPS) clearcache(cachebox,gcfg->cachebox.x*gcfg->cachebox.y);
#else
     float accumweight=0.f;
#endif

     *((float4*)(&p))=n_pos[idx];
     *((float4*)(&v))=n_dir[idx];
     *((float4*)(&f))=n_len[idx];

     gpu_rng_init(t,tnew,n_seed,idx);
     if(gcfg->savedet) clearpath(ppath,gcfg->maxmedia);

     // assuming the initial position is within the domain (mcx_config is supposed to ensure)
     idx1d=gcfg->idx1dorig;
     mediaid=gcfg->mediaidorig;
	  
     if(mediaid==0) {
          return; // the initial position is not within the medium
     }
     *((float4*)(&prop))=gproperty[mediaid];

     /*
      using a while-loop to terminate a thread by np will cause MT RNG to be 3.5x slower
      LL5 RNG will only be slightly slower than for-loop with photon-move criterion
     */
     //while(f.ndone<nphoton) {

     for(np=0;np<nphoton;np++){ // here nphoton actually means photon moves

          GPUDEBUG(("*i= (%d) L=%f w=%e a=%f\n",(int)f.ndone,f.tscat,p.w,f.t));

	  if(f.tscat<=0.f) {  // if this photon has finished the current jump
               rand_need_more(t,tnew);
   	       f.tscat=rand_next_scatlen(t);

               GPUDEBUG(("next scat len=%20.16e \n",f.tscat));
	       if(p.w<1.f){ //weight
                       //random arimuthal angle
                       tmp0=TWO_PI*rand_next_aangle(t); //next arimuth angle
                       sincosf(tmp0,&sphi,&cphi);
                       GPUDEBUG(("next angle phi %20.16e\n",tmp0));

                       //Henyey-Greenstein Phase Function, "Handbook of Optical 
                       //Biomedical Diagnostics",2002,Chap3,p234, also see Boas2002

                       if(prop.g>EPS){  //if prop.g is too small, the distribution of theta is bad
		           tmp0=(1.f-prop.g*prop.g)/(1.f-prop.g+2.f*prop.g*rand_next_zangle(t));
		           tmp0*=tmp0;
		           tmp0=(1.f+prop.g*prop.g-tmp0)/(2.f*prop.g);

                           // when ran=1, CUDA gives me 1.000002 for tmp0 which produces nan later
                           // detected by Ocelot,thanks to Greg Diamos,see http://bit.ly/cR2NMP
                           tmp0=max(-1.f, min(1.f, tmp0));

		           theta=acosf(tmp0);
		           stheta=sinf(theta);
		           ctheta=tmp0;
                       }else{  //Wang1995 has acos(2*ran-1), rather than 2*pi*ran, need to check
			   theta=ONE_PI*rand_next_zangle(t);
                           sincosf(theta,&stheta,&ctheta);
                       }
                       GPUDEBUG(("next scat angle theta %20.16e\n",theta));

		       if( v.z>-1.f+EPS && v.z<1.f-EPS ) {
		           tmp0=1.f-v.z*v.z;   //reuse tmp to minimize registers
		           tmp1=rsqrtf(tmp0);
		           tmp1=stheta*tmp1;
		           *((float4*)(&v))=float4(
				tmp1*(v.x*v.z*cphi - v.y*sphi) + v.x*ctheta,
				tmp1*(v.y*v.z*cphi + v.x*sphi) + v.y*ctheta,
				-tmp1*tmp0*cphi                + v.z*ctheta,
				v.nscat
			   );
                           GPUDEBUG(("new dir: %10.5e %10.5e %10.5e\n",v.x,v.y,v.z));
		       }else{
			   *((float4*)(&v))=float4(stheta*cphi,stheta*sphi,(v.z>0.f)?ctheta:-ctheta,v.nscat);
                           GPUDEBUG(("new dir-z: %10.5e %10.5e %10.5e\n",v.x,v.y,v.z));
 		       }
                       v.nscat++;
	       }
	  }

          n1=prop.n;
	  *((float4*)(&prop))=gproperty[mediaid];
	  len=gcfg->minstep*prop.mus; //Wang1995: gcfg->minstep*(prop.mua+prop.mus)

          p0=p;
	  if(len>f.tscat){  //scattering ends in this voxel: mus*gcfg->minstep > s 
               tmp0=f.tscat/prop.mus;
	       energyabsorbed+=p.w;
   	       *((float4*)(&p))=float4(p.x+v.x*tmp0,p.y+v.y*tmp0,p.z+v.z*tmp0,
                           p.w*expf(-prop.mua*tmp0));
	       energyabsorbed-=p.w;
	       f.tscat=SAME_VOXEL;
	       f.t+=tmp0*prop.n*R_C0;  // accumulative time
               if(gcfg->savedet) ppath[mediaid-1]+=tmp0;
               GPUDEBUG((">>ends in voxel %f<%f %f [%d]\n",f.tscat,len,prop.mus,idx1d));
	  }else{                      //otherwise, move gcfg->minstep
	       energyabsorbed+=p.w;
               if(mediaid!=medid){
                  atten=expf(-prop.mua*gcfg->minstep);
               }
   	       *((float4*)(&p))=float4(p.x+v.x,p.y+v.y,p.z+v.z,p.w*atten);
               medid=mediaid;
	       energyabsorbed-=p.w;
	       f.tscat-=len;     //remaining probability: sum(s_i*mus_i)
	       f.t+=gcfg->minaccumtime*prop.n; //total time
               if(gcfg->savedet) ppath[mediaid-1]+=gcfg->minstep;
               GPUDEBUG((">>keep going %f<%f %f [%d] %e %e\n",f.tscat,len,prop.mus,idx1d,f.t,f.tnext));
	  }

          isdet=(media[idx1d] & DET_MASK);
          idx1dold=idx1d;
          idx1d=(int(floorf(p.z))*gcfg->dimlen.y+int(floorf(p.y))*gcfg->dimlen.x+int(floorf(p.x)));
          GPUDEBUG(("old and new voxel: %d<->%d\n",idx1dold,idx1d));
          if(p.x<0||p.y<0||p.z<0||p.x>=gcfg->maxidx.x||p.y>=gcfg->maxidx.y||p.z>=gcfg->maxidx.z){
	      mediaid=0;
	  }else{
	      mediaid=(media[idx1d] & MED_MASK);
          }

          //if hit the boundary, exceed the max time window or exit the domain, rebound or launch a new one
	  if(mediaid==0||f.t>gcfg->tmax||f.t>gcfg->twin1){
	      float flipdir=0.f;
              float3 htime;            //reflection var

              if(gcfg->doreflect) {
                //time-of-flight to hit the wall in each direction
                htime.x=(v.x>EPS||v.x<-EPS)?(floorf(p0.x)+(v.x>0.f)-p0.x)/v.x:VERY_BIG;
                htime.y=(v.y>EPS||v.y<-EPS)?(floorf(p0.y)+(v.y>0.f)-p0.y)/v.y:VERY_BIG;
                htime.z=(v.z>EPS||v.z<-EPS)?(floorf(p0.z)+(v.z>0.f)-p0.z)/v.z:VERY_BIG;
                //get the direction with the smallest time-of-flight
                tmp0=fminf(fminf(htime.x,htime.y),htime.z);
                flipdir=(tmp0==htime.x?1.f:(tmp0==htime.y?2.f:(tmp0==htime.z&&idx1d!=idx1dold)?3.f:0.f));

                //move to the 1st intersection pt
                tmp0*=JUST_ABOVE_ONE;
                htime.x=floorf(p0.x+tmp0*v.x);
       	        htime.y=floorf(p0.y+tmp0*v.y);
       	        htime.z=floorf(p0.z+tmp0*v.z);

                if(htime.x>=0&&htime.y>=0&&htime.z>=0&&htime.x<gcfg->maxidx.x&&htime.y<gcfg->maxidx.y&&htime.z<gcfg->maxidx.z){
                    if( media[int(htime.z*gcfg->dimlen.y+htime.y*gcfg->dimlen.x+htime.x)]){ //hit again

                     GPUDEBUG((" first try failed: [%.1f %.1f,%.1f] %d (%.1f %.1f %.1f)\n",htime.x,htime.y,htime.z,
                           media[int(htime.z*gcfg->dimlen.y+htime.y*gcfg->dimlen.x+htime.x)], gcfg->maxidx.x, gcfg->maxidx.y,gcfg->maxidx.z));

                     htime.x=(v.x>EPS||v.x<-EPS)?(floorf(p.x)+(v.x<0.f)-p.x)/(-v.x):VERY_BIG;
                     htime.y=(v.y>EPS||v.y<-EPS)?(floorf(p.y)+(v.y<0.f)-p.y)/(-v.y):VERY_BIG;
                     htime.z=(v.z>EPS||v.z<-EPS)?(floorf(p.z)+(v.z<0.f)-p.z)/(-v.z):VERY_BIG;
                     tmp0=fminf(fminf(htime.x,htime.y),htime.z);
                     tmp1=flipdir;   //save the previous ref. interface id
                     flipdir=(tmp0==htime.x?1.f:(tmp0==htime.y?2.f:(tmp0==htime.z&&idx1d!=idx1dold)?3.f:0.f));

                     if(gcfg->doreflect3){
                       tmp0*=JUST_ABOVE_ONE;
                       htime.x=floorf(p.x-tmp0*v.x); //move to the last intersection pt
                       htime.y=floorf(p.y-tmp0*v.y);
                       htime.z=floorf(p.z-tmp0*v.z);

                       if(tmp1!=flipdir&&htime.x>=0&&htime.y>=0&&htime.z>=0&&htime.x<gcfg->maxidx.x&&htime.y<gcfg->maxidx.y&&htime.z<gcfg->maxidx.z){
                           if(! media[int(htime.z*gcfg->dimlen.y+htime.y*gcfg->dimlen.x+htime.x)]){ //this is an air voxel

                               GPUDEBUG((" second try failed: [%.1f %.1f,%.1f] %d (%.1f %.1f %.1f)\n",htime.x,htime.y,htime.z,
                                   media[int(htime.z*gcfg->dimlen.y+htime.y*gcfg->dimlen.x+htime.x)], gcfg->maxidx.x, gcfg->maxidx.y,gcfg->maxidx.z));

                               /*to compute the remaining interface, we used the following fact to accelerate: 
                                 if there exist 3 intersections, photon must pass x/y/z interface exactly once,
                                 we solve the coeff of the following equation to find the last interface:
                                    a*1+b*2+c=3
       	       	       	       	    a*1+b*3+c=2 -> [a b c]=[-1 -1 6], this will give the remaining interface id
       	       	       	       	    a*2+b*3+c=1
                               */
                               flipdir=-tmp1-flipdir+6.f;
                           }
                       }
                     }
                  }
                }
              }

              *((float4*)(&prop))=gproperty[mediaid];

              GPUDEBUG(("->ID%d J%d C%d tlen %e flip %d %.1f!=%.1f dir=%f %f %f pos=%f %f %f\n",idx,(int)v.nscat,
                  (int)f.ndone,f.t, (int)flipdir, n1,prop.n,v.x,v.y,v.z,p.x,p.y,p.z));

              //recycled some old register variables to save memory
	      //if hit boundary within the time window and is n-mismatched, rebound

              if(gcfg->doreflect&&f.t<gcfg->tmax&&f.t<gcfg->twin1&& flipdir>0.f && n1!=prop.n&&p.w>gcfg->minenergy){
	          float Rtotal=1.f;

                  tmp0=n1*n1;
                  tmp1=prop.n*prop.n;
                  if(flipdir>=3.f) { //flip in z axis
                     cphi=fabs(v.z);
                     sphi=v.x*v.x+v.y*v.y;
                     v.z=-v.z;
                  }else if(flipdir>=2.f){ //flip in y axis
                     cphi=fabs(v.y);
       	       	     sphi=v.x*v.x+v.z*v.z;
                     v.y=-v.y;
                  }else if(flipdir>=1.f){ //flip in x axis
                     cphi=fabs(v.x);                //cos(si)
                     sphi=v.y*v.y+v.z*v.z; //sin(si)^2
                     v.x=-v.x;
                  }
		  energyabsorbed+=p.w-p0.w;
                  p=p0;   //move back
                  idx1d=idx1dold;
                  len=1.f-tmp0/tmp1*sphi;   //1-[n1/n2*sin(si)]^2
	          GPUDEBUG((" ref len=%f %f+%f=%f w=%f\n",len,cphi,sphi,cphi*cphi+sphi,p.w));

                  if(len>0.f) { // if not total internal reflection
                     ctheta=tmp0*cphi*cphi+tmp1*len;
                     stheta=2.f*n1*prop.n*cphi*sqrtf(len);
                     Rtotal=(ctheta-stheta)/(ctheta+stheta);
       	       	     ctheta=tmp1*cphi*cphi+tmp0*len;
       	       	     Rtotal=(Rtotal+(ctheta-stheta)/(ctheta+stheta))*0.5f;
	             GPUDEBUG(("  dir=%f %f %f htime=%f %f %f Rs=%f\n",v.x,v.y,v.z,htime.x,htime.y,htime.z,Rtotal));
	             GPUDEBUG(("  ID%d J%d C%d flip=%3f (%d %d) cphi=%f sphi=%f p=%f %f %f p0=%f %f %f\n",
                         idx,(int)v.nscat,(int)f.tnext,
	                 flipdir,idx1dold,idx1d,cphi,sphi,p.x,p.y,p.z,p0.x,p0.y,p0.z));
                  } // else, total internal reflection, no loss
	          if(Rtotal<1.f && rand_next_reflect(t)>Rtotal){
		    	launchnewphoton(&p,&v,&f,&idx1d,&mediaid,isdet,ppath,&energyloss,n_det,detectedphoton);
		  }else{
		        mediaid=(media[idx1d] & MED_MASK);
        	        *((float4*)(&prop))=gproperty[mediaid];
                	n1=prop.n;
	                //v.nscat++;
		  }
              }else{  // launch a new photon
		  launchnewphoton(&p,&v,&f,&idx1d,&mediaid,isdet,ppath,&energyloss,n_det,detectedphoton);
              }
	  }else if(f.t>=f.tnext){
             GPUDEBUG(("field add to %d->%f(%d)  t(%e)>t0(%e)\n",idx1d,p.w,(int)f.ndone,f.t,f.tnext));
             // if t is within the time window, which spans cfg->maxgate*cfg->tstep wide
             if(gcfg->save2pt && f.t>=gcfg->twin0 && f.t<gcfg->twin1){
#ifdef TEST_RACING
                  // enable TEST_RACING to determine how many missing accumulations due to race
                  if( (p.x-gcfg->ps.x)*(p.x-gcfg->ps.x)+(p.y-gcfg->ps.y)*(p.y-gcfg->ps.y)+(p.z-gcfg->ps.z)*(p.z-gcfg->ps.z)>gcfg->skipradius2) {
                      field[idx1d+(int)(floorf((f.t-gcfg->twin0)*gcfg->Rtstep))*gcfg->dimlen.z]+=1.f;
		      cc++;
                  }
#else
  #ifndef USE_ATOMIC
                  // set gcfg->skipradius2 to only start depositing energy when dist^2>gcfg->skipradius2 
                  if(gcfg->skipradius2>EPS){
  #ifdef  USE_CACHEBOX
                      if(p.x<gcfg->cp1.x+1.f && p.x>=gcfg->cp0.x &&
		         p.y<gcfg->cp1.y+1.f && p.y>=gcfg->cp0.y &&
			 p.z<gcfg->cp1.z+1.f && p.z>=gcfg->cp0.z){
                         atomicadd(cachebox+(int(p.z-gcfg->cp0.z)*gcfg->cachebox.y
			      +int(p.y-gcfg->cp0.y)*gcfg->cachebox.x+int(p.x-gcfg->cp0.x)),p.w);
  #else
                      if((p.x-gcfg->ps.x)*(p.x-gcfg->ps.x)+(p.y-gcfg->ps.y)*(p.y-gcfg->ps.y)+(p.z-gcfg->ps.z)*(p.z-gcfg->ps.z)<=gcfg->skipradius2){
                          accumweight+=p.w*prop.mua; // weight*absorption
  #endif
                      }else{
                          field[idx1d+(int)(floorf((f.t-gcfg->twin0)*gcfg->Rtstep))*gcfg->dimlen.z]+=p.w;
                      }
                  }else{
                      field[idx1d+(int)(floorf((f.t-gcfg->twin0)*gcfg->Rtstep))*gcfg->dimlen.z]+=p.w;
                  }
  #else
                  // ifndef CUDA_NO_SM_11_ATOMIC_INTRINSICS
		  atomicadd(& field[idx1d+(int)(floorf((f.t-gcfg->twin0)*gcfg->Rtstep))*gcfg->dimlen.z], p.w);
  #endif
#endif
	     }
             f.tnext+=gcfg->minaccumtime; // fluence is a temporal-integration
	  }
     }
     // cachebox saves the total absorbed energy of all time in the sphere r<sradius.
     // in non-atomic mode, cachebox is more accurate than saving to the grid
     // as it is not influenced by race conditions.
     // now I borrow f.tnext to pass this value back
#ifdef  USE_CACHEBOX
     if(gcfg->skipradius2>EPS){
     	f.tnext=0.f;
        savecache(field,cachebox);
     }
#else
     f.tnext=accumweight;
#endif

     genergy[idx<<1]=energyloss;
     genergy[(idx<<1)+1]=energyabsorbed;

#ifdef TEST_RACING
     n_seed[idx]=cc;
#endif
     n_pos[idx]=*((float4*)(&p));
     n_dir[idx]=*((float4*)(&v));
     n_len[idx]=*((float4*)(&f));
}

kernel void mcx_sum_trueabsorption(float energy[],uchar media[], float field[], int maxgate,uint3 dimlen){
     int i;
     float phi=0.f;
     int idx= blockIdx.x*dimlen.y+blockIdx.y*dimlen.x+ threadIdx.x;

     for(i=0;i<maxgate;i++){
        phi+=field[i*dimlen.z+idx];
     }
     energy[2]+=phi*gproperty[media[idx]].x;
}


/*
   assert cuda memory allocation result
*/
void mcx_cu_assess(hipError_t cuerr,const char *file, const int linenum){
     if(cuerr!=hipSuccess){
         mcx_error(-(int)cuerr,(char *)hipGetErrorString(cuerr),file,linenum);
     }
}


/*
  query GPU info and set active GPU
*/
int mcx_set_gpu(Config *cfg){

#if __DEVICE_EMULATION__
    return 1;
#else
    int dev;
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0){
        fprintf(stderr,"No CUDA-capable GPU device found\n");
        return 0;
    }
    if (cfg->gpuid && cfg->gpuid > deviceCount){
        fprintf(stderr,"Specified GPU ID is out of range\n");
        return 0;
    }
    // scan from the first device
    for (dev = 0; dev<deviceCount; dev++) {
        hipDeviceProp_t dp;
        hipGetDeviceProperties(&dp, dev);
        if (strncmp(dp.name, "Device Emulation", 16)) {
	  if(cfg->isgpuinfo){
	    printf("=============================   GPU Infomation  ================================\n");
	    printf("Device %d of %d:\t\t%s\n",dev+1,deviceCount,dp.name);
	    printf("Compute Capacity:\t%u.%u\n",dp.major,dp.minor);
	    printf("Global Memory:\t\t%u B\nConstant Memory:\t%u B\n\
Shared Memory:\t\t%u B\nRegisters:\t\t%u\nClock Speed:\t\t%.2f GHz\n",
               (unsigned int)dp.totalGlobalMem,(unsigned int)dp.totalConstMem,
               (unsigned int)dp.sharedMemPerBlock,(unsigned int)dp.regsPerBlock,dp.clockRate*1e-6f);
	  #if CUDART_VERSION >= 2000
	       printf("Number of MPs:\t\t%u\nNumber of Cores:\t%u\n",
	          dp.multiProcessorCount,dp.multiProcessorCount<<3);
	  #endif
	  }
          if(cfg->isgpuinfo!=2) break;
	}
    }
    if(cfg->isgpuinfo==2){ //list GPU info only
          exit(0);
    }
    if (cfg->gpuid==0)
        mcx_cu_assess(hipSetDevice(deviceCount-1),__FILE__,__LINE__);
    else
        mcx_cu_assess(hipSetDevice(cfg->gpuid-1),__FILE__,__LINE__);

    return 1;
#endif
}


/*
   master driver code to run MC simulations
*/
void mcx_run_simulation(Config *cfg){

     int i,j,iter;
     float  minstep=MIN(MIN(cfg->steps.x,cfg->steps.y),cfg->steps.z);
     float4 p0=float4(cfg->srcpos.x,cfg->srcpos.y,cfg->srcpos.z,1.f);
     float4 c0=float4(cfg->srcdir.x,cfg->srcdir.y,cfg->srcdir.z,0.f);
     float3 maxidx=float3(cfg->dim.x,cfg->dim.y,cfg->dim.z);
     float t;
     float energyloss=0.f,energyabsorbed=0.f;
     float *energy;
     int threadphoton, oddphotons;

     int photoncount=0,printnum;
     int tic,fieldlen;
     uint3 cp0=cfg->crop0,cp1=cfg->crop1;
     uint2 cachebox;
     uint3 dimlen;
     //uint3 threaddim;
     float Vvox,scale,absorp,eabsorp;

     dim3 mcgrid, mcblock;
     dim3 clgrid, clblock;
     
     int dimxyz=cfg->dim.x*cfg->dim.y*cfg->dim.z;
     
     uchar  *media=(uchar *)(cfg->vol);
     float  *field;
     MCXParam param={cfg->steps,minstep,0,0,cfg->tend,cfg->isrowmajor,
                     cfg->issave2pt,cfg->isreflect,cfg->isref3,cfg->issavedet,0,1.f/cfg->tstep,
		     p0,c0,maxidx,uint3(0,0,0),cp0,cp1,uint2(0,0),cfg->minenergy,
                     cfg->sradius*cfg->sradius,minstep*R_C0,cfg->maxdetphoton,
		     cfg->medianum-1,cfg->detnum,0};

     if(cfg->respin>1){
         field=(float *)calloc(sizeof(float)*dimxyz,cfg->maxgate*2);
     }else{
         field=(float *)calloc(sizeof(float)*dimxyz,cfg->maxgate); //the second half will be used to accumulate
     }
     threadphoton=cfg->nphoton/cfg->nthread/cfg->respin;
     oddphotons=cfg->nphoton-threadphoton*cfg->nthread*cfg->respin;

     float4 *Ppos;
     float4 *Pdir;
     float4 *Plen;
     uint   *Pseed;
     float  *Pdet;
     uint    detected=0,sharedbuf=0;

     if(cfg->nthread%cfg->nblocksize)
     	cfg->nthread=(cfg->nthread/cfg->nblocksize)*cfg->nblocksize;
     mcgrid.x=cfg->nthread/cfg->nblocksize;
     mcblock.x=cfg->nblocksize;

     clgrid.x=cfg->dim.x;
     clgrid.y=cfg->dim.y;
     clblock.x=cfg->dim.z;
	
     Ppos=(float4*)malloc(sizeof(float4)*cfg->nthread);
     Pdir=(float4*)malloc(sizeof(float4)*cfg->nthread);
     Plen=(float4*)malloc(sizeof(float4)*cfg->nthread);
     Pseed=(uint*)malloc(sizeof(uint)*cfg->nthread*RAND_SEED_LEN);
     energy=(float*)calloc(cfg->nthread*2,sizeof(float));
     Pdet=(float*)calloc(cfg->maxdetphoton,sizeof(float)*(cfg->medianum+1));

     uchar *gmedia;
     mcx_cu_assess(hipMalloc((void **) &gmedia, sizeof(uchar)*(dimxyz)),__FILE__,__LINE__);
     float *gfield;
     mcx_cu_assess(hipMalloc((void **) &gfield, sizeof(float)*(dimxyz)*cfg->maxgate),__FILE__,__LINE__);

     //hipBindTexture(0, texmedia, gmedia);

     float4 *gPpos;
     mcx_cu_assess(hipMalloc((void **) &gPpos, sizeof(float4)*cfg->nthread),__FILE__,__LINE__);
     float4 *gPdir;
     mcx_cu_assess(hipMalloc((void **) &gPdir, sizeof(float4)*cfg->nthread),__FILE__,__LINE__);
     float4 *gPlen;
     mcx_cu_assess(hipMalloc((void **) &gPlen, sizeof(float4)*cfg->nthread),__FILE__,__LINE__);
     uint   *gPseed;
     mcx_cu_assess(hipMalloc((void **) &gPseed, sizeof(uint)*cfg->nthread*RAND_SEED_LEN),__FILE__,__LINE__);
     float  *gPdet;
     mcx_cu_assess(hipMalloc((void **) &gPdet, sizeof(float)*cfg->maxdetphoton*(cfg->medianum+1)),__FILE__,__LINE__);
     uint   *gdetected;
     mcx_cu_assess(hipMalloc((void **) &gdetected, sizeof(uint)),__FILE__,__LINE__);

     float *genergy;
     hipMalloc((void **) &genergy, sizeof(float)*cfg->nthread*2);
     
     /*volume is assumbed to be col-major*/
     cachebox.x=(cp1.x-cp0.x+1);
     cachebox.y=(cp1.y-cp0.y+1)*(cp1.x-cp0.x+1);
     dimlen.x=cfg->dim.x;
     dimlen.y=cfg->dim.y*cfg->dim.x;

     dimlen.z=cfg->dim.x*cfg->dim.y*cfg->dim.z;
     param.dimlen=dimlen;
     param.cachebox=cachebox;
     param.idx1dorig=(int(floorf(p0.z))*dimlen.y+int(floorf(p0.y))*dimlen.x+int(floorf(p0.x)));
     param.mediaidorig=(cfg->vol[param.idx1dorig] & MED_MASK);

     /*
      threaddim.x=cfg->dim.z;
      threaddim.y=cfg->dim.y*cfg->dim.z;
      threaddim.z=dimlen.z;
     */
     Vvox=cfg->steps.x*cfg->steps.y*cfg->steps.z;

     if(cfg->seed>0)
     	srand(cfg->seed);
     else
        srand(time(0));
	
     for (i=0; i<cfg->nthread; i++) {
	   Ppos[i]=p0;  // initial position
           Pdir[i]=c0;
           Plen[i]=float4(0.f,0.f,minstep*R_C0,0.f);
     }
     for (i=0; i<cfg->nthread*RAND_SEED_LEN; i++) {
	   Pseed[i]=rand();
     }    
     
     fprintf(cfg->flog,"\
###############################################################################\n\
#                  Monte Carlo Extreme (MCX) -- CUDA                          #\n\
###############################################################################\n\
$MCX $Rev::     $ Last Commit:$Date::                     $ by $Author:: fangq$\n\
###############################################################################\n");

     tic=StartTimer();
#ifdef MCX_TARGET_NAME
     fprintf(cfg->flog,"- variant name: [%s] compiled for GPU Capacity [%d] with CUDA [%d]\n",
             100,MCX_CUDA_ARCH,CUDART_VERSION);
#else
     fprintf(cfg->flog,"- code name: [Vanilla MCX] compiled for GPU Capacity [%d] with CUDA [%d]\n",
             MCX_CUDA_ARCH,CUDART_VERSION);
#endif
     fprintf(cfg->flog,"- compiled with: RNG [%s] Seed Length [%d]\n",MCX_RNG_NAME,RAND_SEED_LEN);
#ifdef SAVE_DETECTORS
     fprintf(cfg->flog,"- this version can save photons at the detectors\n\n");
#else
     fprintf(cfg->flog,"- this version CAN NOT save photons at the detectors\n\n");
#endif
     fprintf(cfg->flog,"threadph=%d oddphotons=%d np=%d nthread=%d repetition=%d\n",threadphoton,oddphotons,
           cfg->nphoton,cfg->nthread,cfg->respin);
     fprintf(cfg->flog,"initializing streams ...\t");
     fflush(cfg->flog);
     fieldlen=dimxyz*cfg->maxgate;

     hipMemcpy(gPpos,  Ppos,  sizeof(float4)*cfg->nthread,  hipMemcpyHostToDevice);
     hipMemcpy(gPdir,  Pdir,  sizeof(float4)*cfg->nthread,  hipMemcpyHostToDevice);
     hipMemcpy(gPlen,  Plen,  sizeof(float4)*cfg->nthread,  hipMemcpyHostToDevice);
     hipMemcpy(gPseed, Pseed, sizeof(uint)  *cfg->nthread*RAND_SEED_LEN,  hipMemcpyHostToDevice);
     hipMemcpy(gfield, field, sizeof(float) *fieldlen, hipMemcpyHostToDevice);
     hipMemcpy(gmedia, media, sizeof(uchar) *dimxyz, hipMemcpyHostToDevice);
     hipMemcpy(genergy,energy,sizeof(float) *cfg->nthread*2, hipMemcpyHostToDevice);
     hipMemcpy(gPdet,  Pdet,  sizeof(float)*cfg->maxdetphoton*(cfg->medianum+1), hipMemcpyHostToDevice);
     hipMemcpy(gdetected,&detected,  sizeof(uint), hipMemcpyHostToDevice);

     hipMemcpyToSymbol(HIP_SYMBOL(gproperty), cfg->prop,  cfg->medianum*sizeof(Medium), 0, hipMemcpyHostToDevice);
     hipMemcpyToSymbol(HIP_SYMBOL(gdetpos), cfg->detpos,  cfg->detnum*sizeof(float4), 0, hipMemcpyHostToDevice);

     fprintf(cfg->flog,"init complete : %d ms\n",GetTimeMillis()-tic);

     /*
         if one has to simulate a lot of time gates, using the GPU global memory
	 requires extra caution. If the total global memory is bigger than the total
	 memory to save all the snapshots, i.e. size(field)*(tend-tstart)/tstep, one
	 simply sets cfg->maxgate to the total gate number; this will run GPU kernel
	 once. If the required memory is bigger than the video memory, set cfg->maxgate
	 to a number which fits, and the snapshot will be saved with an increment of 
	 cfg->maxgate snapshots. In this case, the later simulations will restart from
	 photon launching and exhibit redundancies.
	 
	 The calculation of the energy conservation will only reflect the last simulation.
     */
#ifdef  USE_CACHEBOX
     if(cfg->sradius>EPS)
        sharedbuf+=sizeof(float)*(cachebox.x*cachebox.y);
#endif
     if(cfg->issavedet)
        sharedbuf+=cfg->nblocksize*sizeof(float)*(cfg->medianum-1);

     fprintf(cfg->flog,"requesting %d bytes of shared memory\n",sharedbuf);

     //simulate for all time-gates in maxgate groups per run
     for(t=cfg->tstart;t<cfg->tend;t+=cfg->tstep*cfg->maxgate){

       param.twin0=t;
       param.twin1=t+cfg->tstep*cfg->maxgate;
       hipMemcpyToSymbol(HIP_SYMBOL(gcfg),   &param,     sizeof(MCXParam), 0, hipMemcpyHostToDevice);

       fprintf(cfg->flog,"lauching mcx_main_loop for time window [%.2ens %.2ens] ...\n"
           ,param.twin0*1e9,param.twin1*1e9);

       //total number of repetition for the simulations, results will be accumulated to field
       for(iter=0;iter<cfg->respin;iter++){

           fprintf(cfg->flog,"simulation run#%2d ... \t",iter+1); fflush(cfg->flog);
           mcx_main_loop<<<mcgrid,mcblock,sharedbuf>>>(cfg->nphoton,0,gmedia,gfield,genergy,
	                                               gPseed,gPpos,gPdir,gPlen,gPdet,gdetected);

           hipDeviceSynchronize();
	   hipMemcpy(&detected, gdetected,sizeof(uint),hipMemcpyDeviceToHost);
           fprintf(cfg->flog,"kernel complete:  \t%d ms\nretrieving fields ... \t",GetTimeMillis()-tic);
           mcx_cu_assess(hipGetLastError(),__FILE__,__LINE__);

#ifdef SAVE_DETECTORS
           if(cfg->issavedet){
           	hipMemcpy(Pdet, gPdet,sizeof(float)*cfg->maxdetphoton*(cfg->medianum+1),hipMemcpyDeviceToHost);
	        mcx_cu_assess(hipGetLastError(),__FILE__,__LINE__);
		if(detected>cfg->maxdetphoton){
			fprintf(cfg->flog,"WARNING: the detected photon (%d) \
is more than what your have specified (%d), please use the -H option to specify a greater number\t"
                           ,detected,cfg->maxdetphoton);
		}else{
			fprintf(cfg->flog,"detected %d photons\t",detected);
		}
		//cfg->his.totalphoton=
		cfg->his.unitinmm=cfg->unitinmm;
		cfg->his.detected=detected;
		cfg->his.savedphoton=MIN(detected,cfg->maxdetphoton);
		mcx_savedata(Pdet,cfg->his.savedphoton*(cfg->medianum+1),t>cfg->tstart,"mch",cfg);
	   }
#endif
	   //handling the 2pt distributions
           if(cfg->issave2pt){
               hipMemcpy(field, gfield,sizeof(float) *dimxyz*cfg->maxgate,hipMemcpyDeviceToHost);
               fprintf(cfg->flog,"transfer complete:\t%d ms\n",GetTimeMillis()-tic);  fflush(cfg->flog);

               if(cfg->respin>1){
                   for(i=0;i<fieldlen;i++)  //accumulate field, can be done in the GPU
                      field[fieldlen+i]+=field[i];
               }
               if(iter+1==cfg->respin){ 
                   if(cfg->respin>1)  //copy the accumulated fields back
                       memcpy(field,field+fieldlen,sizeof(float)*fieldlen);

                   if(cfg->isnormalized){
                       //normalize field if it is the last iteration, temporarily do it in CPU
                       //mcx_sum_trueabsorption<<<clgrid,clblock>>>(genergy,gmedia,gfield,
                       //  	cfg->maxgate,threaddim);

                       fprintf(cfg->flog,"normizing raw data ...\t");

                       hipMemcpy(energy,genergy,sizeof(float)*cfg->nthread*2,hipMemcpyDeviceToHost);
		       hipMemcpy(Plen,  gPlen,  sizeof(float4)*cfg->nthread, hipMemcpyDeviceToHost);
                       eabsorp=0.f;
                       for(i=1;i<cfg->nthread;i++){
                           energy[0]+=energy[i<<1];
       	       	       	   energy[1]+=energy[(i<<1)+1];
                           eabsorp+=Plen[i].z;  // the accumulative absorpted energy near the source
                       }
       	       	       for(i=0;i<dimxyz;i++){
                           absorp=0.f;
                           for(j=0;j<cfg->maxgate;j++)
                              absorp+=field[j*dimxyz+i];
                           eabsorp+=absorp*cfg->prop[media[i]].mua;
       	       	       }
                       scale=energy[1]/((energy[0]+energy[1])*Vvox*cfg->tstep*eabsorp);
		       if(cfg->unitinmm!=1.f) 
		          scale*=(cfg->unitinmm*cfg->unitinmm);
                       fprintf(cfg->flog,"normalization factor alpha=%f\n",scale);  fflush(cfg->flog);
                       mcx_normalize(field,scale,fieldlen);
                   }
                   fprintf(cfg->flog,"data normalization complete : %d ms\n",GetTimeMillis()-tic);

                   fprintf(cfg->flog,"saving data to file ...\t");
                   mcx_savedata(field,fieldlen,t>cfg->tstart,"mc2",cfg);
                   fprintf(cfg->flog,"saving data complete : %d ms\n\n",GetTimeMillis()-tic);
                   fflush(cfg->flog);
               }
           }
	   //initialize the next simulation
	   if(param.twin1<cfg->tend && iter<cfg->respin){
                  hipMemset(gfield,0,sizeof(float)*fieldlen); // cost about 1 ms
                  hipMemset(gPdet,0,sizeof(float)*cfg->maxdetphoton*(cfg->medianum+1));
                  hipMemset(gdetected,0,sizeof(float));

 		  hipMemcpy(gPpos,  Ppos,  sizeof(float4)*cfg->nthread,  hipMemcpyHostToDevice); //following 3 cost about 50 ms
		  hipMemcpy(gPdir,  Pdir,  sizeof(float4)*cfg->nthread,  hipMemcpyHostToDevice);
		  hipMemcpy(gPlen,  Plen,  sizeof(float4)*cfg->nthread,  hipMemcpyHostToDevice);
	   }
	   if(cfg->respin>1 && RAND_SEED_LEN>1){
               for (i=0; i<cfg->nthread*RAND_SEED_LEN; i++)
		   Pseed[i]=rand();
	       hipMemcpy(gPseed, Pseed, sizeof(uint)*cfg->nthread*RAND_SEED_LEN,  hipMemcpyHostToDevice);
	   }
       }
       if(param.twin1<cfg->tend){
            hipMemset(genergy,0,sizeof(float)*cfg->nthread*2);
       }
     }

     hipMemcpy(Ppos,  gPpos, sizeof(float4)*cfg->nthread, hipMemcpyDeviceToHost);
     hipMemcpy(Pdir,  gPdir, sizeof(float4)*cfg->nthread, hipMemcpyDeviceToHost);
     hipMemcpy(Plen,  gPlen, sizeof(float4)*cfg->nthread, hipMemcpyDeviceToHost);
     hipMemcpy(Pseed, gPseed,sizeof(uint)  *cfg->nthread*RAND_SEED_LEN,   hipMemcpyDeviceToHost);
     hipMemcpy(energy,genergy,sizeof(float)*cfg->nthread*2,hipMemcpyDeviceToHost);

     for (i=0; i<cfg->nthread; i++) {
	  photoncount+=(int)Plen[i].w;
          energyloss+=energy[i<<1];
          energyabsorbed+=energy[(i<<1)+1];
     }

#ifdef TEST_RACING
     {
       float totalcount=0.f,hitcount=0.f;
       for (i=0; i<fieldlen; i++)
          hitcount+=field[i];
       for (i=0; i<cfg->nthread; i++)
	  totalcount+=Pseed[i];
     
       fprintf(cfg->flog,"expected total recording number: %f, got %f, missed %f\n",
          totalcount,hitcount,(totalcount-hitcount)/totalcount);
     }
#endif

     printnum=cfg->nthread<cfg->printnum?cfg->nthread:cfg->printnum;
     for (i=0; i<printnum; i++) {
           fprintf(cfg->flog,"% 4d[A% f % f % f]C%3d J%5d W% 8f(P%6.3f %6.3f %6.3f)T% 5.3e L% 5.3f %.0f\n", i,
            Pdir[i].x,Pdir[i].y,Pdir[i].z,(int)Plen[i].w,(int)Pdir[i].w,Ppos[i].w, 
            Ppos[i].x,Ppos[i].y,Ppos[i].z,Plen[i].y,Plen[i].x,(float)Pseed[i]);
     }
     // total energy here equals total simulated photons+unfinished photons for all threads
     fprintf(cfg->flog,"simulated %d photons (%d) with %d threads (repeat x%d)\n",
             photoncount,cfg->nphoton,cfg->nthread,cfg->respin); fflush(cfg->flog);
     fprintf(cfg->flog,"exit energy:%16.8e + absorbed energy:%16.8e = total: %16.8e\n",
             energyloss,energyabsorbed,energyloss+energyabsorbed);fflush(cfg->flog);
     fflush(cfg->flog);

     hipFree(gmedia);
     hipFree(gfield);
     hipFree(gPpos);
     hipFree(gPdir);
     hipFree(gPlen);
     hipFree(gPseed);
     hipFree(genergy);
     hipFree(gPdet);
     hipFree(gdetected);

     free(Ppos);
     free(Pdir);
     free(Plen);
     free(Pseed);
     free(Pdet);
     free(energy);
     free(field);
}
