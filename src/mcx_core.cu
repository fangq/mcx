#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
//
//  Monte Carlo eXtreme (MCX)  - GPU accelerated 3D Monte Carlo transport simulation
//  Author: Qianqian Fang <q.fang at neu.edu>
//
//  Reference (Fang2009):
//        Qianqian Fang and David A. Boas, "Monte Carlo Simulation of Photon 
//        Migration in 3D Turbid Media Accelerated by Graphics Processing 
//        Units," Optics Express, vol. 17, issue 22, pp. 20178-20190 (2009)
//
//  mcx_core.cu: GPU kernels and CUDA host code
//
//  License: GNU General Public License v3, see LICENSE.txt for details
//
////////////////////////////////////////////////////////////////////////////////

#include "br2cu.h"
#include "mcx_core.h"
#include "tictoc.h"
#include "mcx_const.h"

#if defined(USE_XORSHIFT128P_RAND)
    #include "xorshift128p_rand.cu" // use xorshift128+ RNG (XORSHIFT128P)
#elif defined(USE_POSIX_RAND)
    #include "posix_rand.cu"        // use POSIX erand48 RNG (POSIX)
#elif defined(USE_MT_RAND)
    #include "mt_rand_s.cu"         // use Mersenne Twister RNG (MT), depreciated
#else
    #include "logistic_rand.cu"     // use Logistic Lattice ring 5 RNG (LL5)
#endif

#ifdef _OPENMP
    #include <omp.h>
#endif

#define CUDA_ASSERT(a)      mcx_cu_assess((a),__FILE__,__LINE__)

// optical properties saved in the constant memory
// {x}:mua,{y}:mus,{z}:anisotropy (g),{w}:refractive index (n)
__constant__ float4 gproperty[MAX_PROP];

__constant__ float4 gdetpos[MAX_DETECTORS];

// kernel constant parameters
__constant__ MCXParam gcfg[1];

__device__ uint gjumpdebug[1];

extern __shared__ float sharedmem[]; //max 64 tissue types when block size=64

// tested with texture memory for media, only improved 1% speed
// to keep code portable, use global memory for now
// also need to change all media[idx1d] to tex1Dfetch() below
//texture<uchar, 1, hipReadModeElementType> texmedia;

__device__ inline void atomicadd(float* address, float value){

#if __CUDA_ARCH__ >= 200 // for Fermi, atomicAdd supports floats

  atomicAdd(address,value);

#elif __CUDA_ARCH__ >= 110

// float-atomic-add from 
// http://forums.nvidia.com/index.php?showtopic=158039&view=findpost&p=991561
  float old = value;  
  while ((old = atomicExch(address, atomicExch(address, 0.0f)+old))!=0.0f);

#endif

}

__device__ inline void clearpath(float *p,int maxmediatype){
      uint i;
      for(i=0;i<maxmediatype;i++)
      	   p[i]=0.f;
}

__device__ inline void clearcache(float *p,int len){
      uint i;
      if(threadIdx.x==0)
        for(i=0;i<len;i++)
      	   p[i]=0.f;
}

#ifdef  USE_CACHEBOX
__device__ inline void savecache(float *data,float *cache){
      uint x,y,z;
      if(threadIdx.x==0){
        for(z=gcfg->cp0.z;z<=gcfg->cp1.z;z++)
           for(y=gcfg->cp0.y;y<=gcfg->cp1.y;y++)
              for(x=gcfg->cp0.x;x<=gcfg->cp1.x;x++){
                 atomicadd(data+z*gcfg->dimlen.y+y*gcfg->dimlen.x+x,
		    cache[(z-gcfg->cp0.z)*gcfg->cachebox.y+(y-gcfg->cp0.y)*gcfg->cachebox.x+(x-gcfg->cp0.x)]);
	      }
      }
}
#endif

#ifdef SAVE_DETECTORS
__device__ inline uint finddetector(MCXpos *p0){
      uint i;
      for(i=0;i<gcfg->detnum;i++){
      	if((gdetpos[i].x-p0->x)*(gdetpos[i].x-p0->x)+
	   (gdetpos[i].y-p0->y)*(gdetpos[i].y-p0->y)+
	   (gdetpos[i].z-p0->z)*(gdetpos[i].z-p0->z) < gdetpos[i].w*gdetpos[i].w){
	        return i+1;
	   }
      }
      return 0;
}

__device__ inline void savedetphoton(float n_det[],uint *detectedphoton,float nscat,float *ppath,MCXpos *p0,RandType t[RAND_BUF_LEN],RandType *seeddata){
      uint detid;
      detid=finddetector(p0);
      if(detid){
	 uint baseaddr=atomicAdd(detectedphoton,1);
	 if(baseaddr<gcfg->maxdetphoton){
	    uint i;
	    for(i=0;i<gcfg->issaveseed*RAND_BUF_LEN;i++)
	        seeddata[baseaddr*RAND_BUF_LEN+i]=t[i]; // save photon seed for replay
	    baseaddr*=gcfg->maxmedia+2;
	    n_det[baseaddr++]=detid;
	    n_det[baseaddr++]=nscat;
	    for(i=0;i<gcfg->maxmedia;i++)
		n_det[baseaddr+i]=ppath[i]; // save partial pathlength to the memory
	 }
      }
}
#endif
__device__ inline void savedebugdata(MCXpos *p,uint id,float *gdebugdata){
      uint pos=atomicAdd(gjumpdebug,1);
      if(pos<gcfg->maxjumpdebug){
         pos*=MCX_DEBUG_REC_LEN;
         ((uint *)gdebugdata)[pos++]=id;
         gdebugdata[pos++]=p->x;
         gdebugdata[pos++]=p->y;
         gdebugdata[pos++]=p->z;
         gdebugdata[pos++]=p->w;
         gdebugdata[pos++]=0;
      }
}
__device__ inline float mcx_nextafterf(float a, int dir){
      union{
          float f;
	  uint  i;
      } num;
      num.f=a+gcfg->maxvoidstep;
      num.i+=dir ^ (num.i & 0x80000000U);
      return num.f-gcfg->maxvoidstep;
}

__device__ inline float hitgrid(float3 *p0, float3 *v, float *htime,float* rv,int *id){
      float dist;

      //time-of-flight to hit the wall in each direction
      htime[0]=fabs((floorf(p0->x)+(v->x>0.f)-p0->x)*rv[0]); // absolute distance of travel in x/y/z
      htime[1]=fabs((floorf(p0->y)+(v->y>0.f)-p0->y)*rv[1]);
      htime[2]=fabs((floorf(p0->z)+(v->z>0.f)-p0->z)*rv[2]);

      //get the direction with the smallest time-of-flight
      dist=fminf(fminf(htime[0],htime[1]),htime[2]);
      (*id)=(dist==htime[0]?0:(dist==htime[1]?1:2));

      //p0 is inside, p is outside, move to the 1st intersection pt, now in the air side, to be corrected in the else block
      htime[0]=p0->x+dist*v->x;
      htime[1]=p0->y+dist*v->y;
      htime[2]=p0->z+dist*v->z;

      (*id==0) ?
          (htime[0]=mcx_nextafterf(__float2int_rn(htime[0]), (v->x > 0.f)-(v->x < 0.f))) :
	  ((*id==1) ? 
	      (htime[1]=mcx_nextafterf(__float2int_rn(htime[1]), (v->y > 0.f)-(v->y < 0.f))) :
	      (htime[2]=mcx_nextafterf(__float2int_rn(htime[2]), (v->z > 0.f)-(v->z < 0.f))) );

      return dist;
}

__device__ inline void transmit(MCXdir *v, float n1, float n2,int flipdir){
      float tmp0=n1/n2;
      v->x*=tmp0;
      v->y*=tmp0;
      v->z*=tmp0;
      (flipdir==0) ?
          (v->x=sqrtf(1.f - v->y*v->y - v->z*v->z)*((v->x>0.f)-(v->x<0.f))):
	  ((flipdir==1) ? 
	      (v->y=sqrtf(1.f - v->x*v->x - v->z*v->z)*((v->y>0.f)-(v->y<0.f))):
	      (v->z=sqrtf(1.f - v->x*v->x - v->y*v->y)*((v->z>0.f)-(v->z<0.f))));
}

__device__ inline float reflectcoeff(MCXdir *v, float n1, float n2, int flipdir){
      float Icos=fabs((flipdir==0) ? v->x : (flipdir==1 ? v->y : v->z));
      float tmp0=n1*n1;
      float tmp1=n2*n2;
      float tmp2=1.f-tmp0/tmp1*(1.f-Icos*Icos); /*1-[n1/n2*sin(si)]^2 = cos(ti)^2*/
      if(tmp2>0.f){ // partial reflection
          float Re,Im,Rtotal;
	  Re=tmp0*Icos*Icos+tmp1*tmp2;
	  tmp2=sqrtf(tmp2); /*to save one sqrt*/
	  Im=2.f*n1*n2*Icos*tmp2;
	  Rtotal=(Re-Im)/(Re+Im);     /*Rp*/
	  Re=tmp1*Icos*Icos+tmp0*tmp2*tmp2;
	  Rtotal=(Rtotal+(Re-Im)/(Re+Im))*0.5f; /*(Rp+Rs)/2*/
	  return Rtotal;
      }else{ // total reflection
          return 1.f;
      }
}

/* if the source location is outside of the volume or 
in an void voxel, mcx advances the photon in v.{xyz} direction
until it hits an non-zero voxel */
__device__ inline int skipvoid(MCXpos *p,MCXdir *v,MCXtime *f,float3* rv,uchar media[]){
      int count=1,idx1d;
      while(1){
          if(p->x>=0.f && p->y>=0.f && p->z>=0.f && p->x < gcfg->maxidx.x
               && p->y < gcfg->maxidx.y && p->z < gcfg->maxidx.z){
	    idx1d=(int(floorf(p->z))*gcfg->dimlen.y+int(floorf(p->y))*gcfg->dimlen.x+int(floorf(p->x)));
	    if(media[idx1d] & MED_MASK){ // if inside
                GPUDEBUG(("inside volume [%f %f %f] v=<%f %f %f>\n",p->x,p->y,p->z,v->x,v->y,v->z));
	        float3 htime;
                int flipdir;
                p->x-=v->x;
                p->y-=v->y;
                p->z-=v->z;
                f->t-=gcfg->minaccumtime;
                idx1d=(int(floorf(p->z))*gcfg->dimlen.y+int(floorf(p->y))*gcfg->dimlen.x+int(floorf(p->x)));

                GPUDEBUG(("look for entry p0=[%f %f %f] rv=[%f %f %f]\n",p->x,p->y,p->z,rv->x,rv->y,rv->z));
		count=0;
		while(!(p->x>=0.f && p->y>=0.f && p->z>=0.f && p->x < gcfg->maxidx.x
                  && p->y < gcfg->maxidx.y && p->z < gcfg->maxidx.z) || !(media[idx1d] & MED_MASK)){ // at most 3 times
	            f->t+=gcfg->minaccumtime*hitgrid((float3*)p,(float3*)v,&htime.x,&rv->x,&flipdir);
                    *((float4*)(p))=float4(htime.x,htime.y,htime.z,p->w);
                    idx1d=(int(floorf(p->z))*gcfg->dimlen.y+int(floorf(p->y))*gcfg->dimlen.x+int(floorf(p->x)));
                    GPUDEBUG(("entry p=[%f %f %f] flipdir=%d\n",p->x,p->y,p->z,flipdir));

		    if(count++>3){
		       GPUDEBUG(("fail to find entry point after 3 iterations, something is wrong, abort!!"));
		       break;
		    }
		}
                f->t= (gcfg->voidtime) ? f->t : 0.f;

		if(gproperty[media[idx1d] & MED_MASK].w!=gproperty[0].w){
	            p->w*=1.f-reflectcoeff(v, gproperty[0].w,gproperty[media[idx1d] & MED_MASK].w,flipdir);
                    GPUDEBUG(("transmitted intensity w=%e\n",p->w));
	            if(p->w>EPS){
		        transmit(v, gproperty[0].w,gproperty[media[idx1d] & MED_MASK].w,flipdir);
                        GPUDEBUG(("transmit into volume v=<%f %f %f>\n",v->x,v->y,v->z));
                    }
		}
		return idx1d;
	    }
          }
	  if( (p->x<0.f) && (v->x<=0.f) || (p->x >= gcfg->maxidx.x) && (v->x>=0.f)
	   || (p->y<0.f) && (v->y<=0.f) || (p->y >= gcfg->maxidx.y) && (v->y>=0.f)
	   || (p->z<0.f) && (v->z<=0.f) || (p->z >= gcfg->maxidx.z) && (v->z>=0.f))
	      return -1;
	  *((float4*)(p))=float4(p->x+v->x,p->y+v->y,p->z+v->z,p->w);
          GPUDEBUG(("inside void [%f %f %f]\n",p->x,p->y,p->z));
          f->t+=gcfg->minaccumtime;
	  if(count++>gcfg->maxvoidstep)
	      return -1;
      }
}


__device__ inline void rotatevector(MCXdir *v, float stheta, float ctheta, float sphi, float cphi){
      if( v->z>-1.f+EPS && v->z<1.f-EPS ) {
   	  float tmp0=1.f-v->z*v->z;
   	  float tmp1=stheta*rsqrtf(tmp0);
   	  *((float4*)v)=float4(
   	       tmp1*(v->x*v->z*cphi - v->y*sphi) + v->x*ctheta,
   	       tmp1*(v->y*v->z*cphi + v->x*sphi) + v->y*ctheta,
   	      -tmp1*tmp0*cphi                    + v->z*ctheta,
   	       v->nscat
   	  );
      }else{
   	  *((float4*)v)=float4(stheta*cphi,stheta*sphi,(v->z>0.f)?ctheta:-ctheta,v->nscat);
      }
      GPUDEBUG(("new dir: %10.5e %10.5e %10.5e\n",v->x,v->y,v->z));
}


template <int mcxsource>
__device__ inline int launchnewphoton(MCXpos *p,MCXdir *v,MCXtime *f,float3* rv,Medium *prop,uint *idx1d,
           uint *mediaid,float *w0,float *Lmove,uint isdet, float ppath[],float energyloss[],float energylaunched[],float n_det[],uint *dpnum,
	   RandType t[RAND_BUF_LEN],RandType photonseed[RAND_BUF_LEN],
	   uchar media[],float srcpattern[],int threadid,RandType rngseed[],RandType seeddata[],float gdebugdata[],volatile int gprogress[]){
      int launchattempt=1;

      if(p->w>=0.f){
          *energyloss+=p->w;  // sum all the remaining energy
#ifdef SAVE_DETECTORS
      // let's handle detectors here
          if(gcfg->savedet){
             if(isdet && *mediaid==0)
	         savedetphoton(n_det,dpnum,v->nscat,ppath,p,photonseed,seeddata);
             clearpath(ppath,gcfg->maxmedia);
          }
#endif
      }

      if((int)(f->ndone)>=(gcfg->threadphoton+(threadid<gcfg->oddphotons))){
          return 1; // all photos complete
      }
      if(gcfg->seed==SEED_FROM_FILE){
          int seedoffset=(threadid*gcfg->threadphoton+min(threadid,gcfg->oddphotons-1)+(int)f->ndone)*RAND_BUF_LEN;
          for(int i=0;i<RAND_BUF_LEN;i++)
	      t[i]=rngseed[seedoffset+i];
      }
      do{
	  *((float4*)p)=gcfg->ps;
	  *((float4*)v)=gcfg->c0;
	  *((float4*)f)=float4(0.f,0.f,gcfg->minaccumtime,f->ndone);
          *idx1d=gcfg->idx1dorig;
          *mediaid=gcfg->mediaidorig;
	  if(gcfg->issaveseed)
              copystate(t,photonseed);
	  
	  switch(mcxsource) {
		case(MCX_SRC_PLANAR):
		case(MCX_SRC_PATTERN):
		case(MCX_SRC_FOURIER): { /*a rectangular grid over a plane*/
		      float rx=rand_uniform01(t);
		      float ry=rand_uniform01(t);
		      *((float4*)p)=float4(p->x+rx*gcfg->srcparam1.x+ry*gcfg->srcparam2.x,
					   p->y+rx*gcfg->srcparam1.y+ry*gcfg->srcparam2.y,
					   p->z+rx*gcfg->srcparam1.z+ry*gcfg->srcparam2.z,
					   p->w);
		      if(gcfg->srctype==MCX_SRC_PATTERN) // need to prevent rx/ry=1 here
			  p->w=srcpattern[(int)(ry*JUST_BELOW_ONE*gcfg->srcparam2.w)*(int)(gcfg->srcparam1.w)+(int)(rx*JUST_BELOW_ONE*gcfg->srcparam1.w)];
		      else if(gcfg->srctype==MCX_SRC_FOURIER){
			  p->w=(cosf((floorf(gcfg->srcparam1.w)*rx+floorf(gcfg->srcparam2.w)*ry
				  +gcfg->srcparam1.w-floorf(gcfg->srcparam1.w))*TWO_PI)*(1.f-gcfg->srcparam2.w+floorf(gcfg->srcparam2.w))+1.f)*0.5f; //between 0 and 1
		      }
		      *idx1d=(int(floorf(p->z))*gcfg->dimlen.y+int(floorf(p->y))*gcfg->dimlen.x+int(floorf(p->x)));
		      if(p->x<0.f || p->y<0.f || p->z<0.f || p->x>=gcfg->maxidx.x || p->y>=gcfg->maxidx.y || p->z>=gcfg->maxidx.z){
			  *mediaid=0;
		      }else{
			  *mediaid=media[*idx1d];
		      }
		      break;
		}
		case(MCX_SRC_FOURIERX):
		case(MCX_SRC_FOURIERX2D): { // [v1x][v1y][v1z][|v2|]; [kx][ky][phi0][M], unit(v0) x unit(v1)=unit(v2)
		      float rx=rand_uniform01(t);
		      float ry=rand_uniform01(t);
		      float4 v2=gcfg->srcparam1;
		      // calculate v2 based on v2=|v2| * unit(v0) x unit(v1)
		      v2.w*=rsqrt(gcfg->srcparam1.x*gcfg->srcparam1.x+gcfg->srcparam1.y*gcfg->srcparam1.y+gcfg->srcparam1.z*gcfg->srcparam1.z);
		      v2.x=v2.w*(gcfg->c0.y*gcfg->srcparam1.z - gcfg->c0.z*gcfg->srcparam1.y);
		      v2.y=v2.w*(gcfg->c0.z*gcfg->srcparam1.x - gcfg->c0.x*gcfg->srcparam1.z); 
		      v2.z=v2.w*(gcfg->c0.x*gcfg->srcparam1.y - gcfg->c0.y*gcfg->srcparam1.x);
		      *((float4*)p)=float4(p->x+rx*gcfg->srcparam1.x+ry*v2.x,
					   p->y+rx*gcfg->srcparam1.y+ry*v2.y,
					   p->z+rx*gcfg->srcparam1.z+ry*v2.z,
					   p->w);
		      if(gcfg->srctype==MCX_SRC_FOURIERX2D)
			 p->w=(sinf((gcfg->srcparam2.x*rx+gcfg->srcparam2.z)*TWO_PI)*sinf((gcfg->srcparam2.y*ry+gcfg->srcparam2.w)*TWO_PI)+1.f)*0.5f; //between 0 and 1
		      else
			 p->w=(cosf((gcfg->srcparam2.x*rx+gcfg->srcparam2.y*ry+gcfg->srcparam2.z)*TWO_PI)*(1.f-gcfg->srcparam2.w)+1.f)*0.5f; //between 0 and 1
   
		      *idx1d=(int(floorf(p->z))*gcfg->dimlen.y+int(floorf(p->y))*gcfg->dimlen.x+int(floorf(p->x)));
		      if(p->x<0.f || p->y<0.f || p->z<0.f || p->x>=gcfg->maxidx.x || p->y>=gcfg->maxidx.y || p->z>=gcfg->maxidx.z){
			  *mediaid=0;
		      }else{
			  *mediaid=media[*idx1d];
		      }
		      break;
		}
		case(MCX_SRC_DISK):
		case(MCX_SRC_GAUSSIAN): { // uniform disk distribution or Gaussian-beam
		      // Uniform disk point picking
		      // http://mathworld.wolfram.com/DiskPointPicking.html
		      float sphi, cphi;
		      float phi=TWO_PI*rand_uniform01(t);
		      sincosf(phi,&sphi,&cphi);
		     float r;
		     if(gcfg->srctype==MCX_SRC_DISK)
			 r=sqrtf(rand_uniform01(t))*gcfg->srcparam1.x;
		     else
			 r=sqrtf(-logf(rand_uniform01(t)))*gcfg->srcparam1.x;

		      if( v->z>-1.f+EPS && v->z<1.f-EPS ) {
			  float tmp0=1.f-v->z*v->z;
			  float tmp1=r*rsqrtf(tmp0);
			  *((float4*)p)=float4(
			       p->x+tmp1*(v->x*v->z*cphi - v->y*sphi),
			       p->y+tmp1*(v->y*v->z*cphi + v->x*sphi),
			       p->z-tmp1*tmp0*cphi                   ,
			       p->w
			  );
			  GPUDEBUG(("new dir: %10.5e %10.5e %10.5e\n",v->x,v->y,v->z));
		      }else{
			  p->x+=r*cphi;
			  p->y+=r*sphi;
			  GPUDEBUG(("new dir-z: %10.5e %10.5e %10.5e\n",v->x,v->y,v->z));
		      }
		      *idx1d=(int(floorf(p->z))*gcfg->dimlen.y+int(floorf(p->y))*gcfg->dimlen.x+int(floorf(p->x)));
		      if(p->x<0.f || p->y<0.f || p->z<0.f || p->x>=gcfg->maxidx.x || p->y>=gcfg->maxidx.y || p->z>=gcfg->maxidx.z){
			  *mediaid=0;
		      }else{
			  *mediaid=media[*idx1d];
		      }
		      break;
		  }
		case(MCX_SRC_CONE):
		case(MCX_SRC_ISOTROPIC):
		case(MCX_SRC_ARCSINE): {
		      // Uniform point picking on a sphere 
		      // http://mathworld.wolfram.com/SpherePointPicking.html
		      float ang,stheta,ctheta,sphi,cphi;
		      ang=TWO_PI*rand_uniform01(t); //next arimuth angle
		      sincosf(ang,&sphi,&cphi);
		      if(gcfg->srctype==MCX_SRC_CONE){  // a solid-angle section of a uniform sphere
			  do{
			      ang=(gcfg->srcparam1.y>0) ? TWO_PI*rand_uniform01(t) : acosf(2.f*rand_uniform01(t)-1.f); //sine distribution
			  }while(ang>gcfg->srcparam1.x);
		      }else{
			  if(gcfg->srctype==MCX_SRC_ISOTROPIC) // uniform sphere
			      ang=acosf(2.f*rand_uniform01(t)-1.f); //sine distribution
			  else
			      ang=ONE_PI*rand_uniform01(t); //uniform distribution in zenith angle, arcsine
		      }
		      sincosf(ang,&stheta,&ctheta);
		      rotatevector(v,stheta,ctheta,sphi,cphi);
		      break;
		}
		case(MCX_SRC_ZGAUSSIAN): {
		      float ang,stheta,ctheta,sphi,cphi;
		      ang=TWO_PI*rand_uniform01(t); //next arimuth angle
		      sincosf(ang,&sphi,&cphi);
		      ang=sqrtf(-2.f*logf(rand_uniform01(t)))*(1.f-2.f*rand_uniform01(t))*gcfg->srcparam1.x;
		      sincosf(ang,&stheta,&ctheta);
		      rotatevector(v,stheta,ctheta,sphi,cphi);
		      break;
		}
		case(MCX_SRC_LINE):
		case(MCX_SRC_SLIT): {
		      float r=rand_uniform01(t);
		      *((float4*)p)=float4(p->x+r*gcfg->srcparam1.x,
					   p->y+r*gcfg->srcparam1.y,
					   p->z+r*gcfg->srcparam1.z,
					   p->w);
		      if(gcfg->srctype==MCX_SRC_LINE){
			      float s,p;
			      r=1.f-2.f*rand_uniform01(t);
			      s=1.f-2.f*rand_uniform01(t);
			      p=sqrt(1.f-v->x*v->x-v->y*v->y)*(rand_uniform01(t)>0.5f ? 1.f : -1.f);
			      *((float4*)v)=float4(v->y*p-v->z*s,v->z*r-v->x*p,v->x*s-v->y*r,v->nscat);
		      }
		      break;
		}
	  }
	
	    *rv=float3(__fdividef(1.f,v->x),__fdividef(1.f,v->y),__fdividef(1.f,v->z));
	  if((*mediaid & MED_MASK)==0){
             int idx=skipvoid(p, v, f, rv, media); //specular reflection of the bbx is taken care of here
             if(idx>=0){
		 *idx1d=idx;
		 *mediaid=media[*idx1d];
	     }
	  }
	  
	  if(launchattempt++>gcfg->maxvoidstep)
	     return -1;  // launch failed
      }while((*mediaid & MED_MASK)==0 || p->w<=gcfg->minenergy);
      f->ndone++; // launch successfully
      *((float4*)(prop))=gproperty[*mediaid & MED_MASK]; //always use mediaid to read gproperty[]
      if(gcfg->debuglevel & MCX_DEBUG_MOVE)
          savedebugdata(p,(uint)f->ndone+threadid*gcfg->threadphoton+umin(threadid,(threadid<gcfg->oddphotons)*threadid),gdebugdata);

      /*total energy enters the volume. for diverging/converting 
      beams, this is less than nphoton due to specular reflection 
      loss. This is different from the wide-field MMC, where the 
      total launched energy includes the specular reflection loss*/
      
      *energylaunched+=p->w;
      *w0=p->w;
      *Lmove=0.f;
      if((gcfg->debuglevel & MCX_DEBUG_PROGRESS) && ((int)(f->ndone) & 1) && (threadid==0 || threadid==blockDim.x * gridDim.x - 1 
          || threadid==((blockDim.x * gridDim.x)>>1))) // use the 1st, middle and last thread for progress report
          gprogress[0]++;
      return 0;
}

kernel void mcx_test_rng(float field[],uint n_seed[]){
     int idx= blockDim.x * blockIdx.x + threadIdx.x;
     int i;
     int len=gcfg->maxidx.x*gcfg->maxidx.y*gcfg->maxidx.z*(int)((gcfg->twin1-gcfg->twin0)*gcfg->Rtstep+0.5f);
     RandType t[RAND_BUF_LEN];

     gpu_rng_init(t,n_seed,idx);

     for(i=0;i<len;i++){
	   field[i]=rand_uniform01(t);
     }
}

/**
   this is the core Monte Carlo simulation kernel, please see Fig. 1 in Fang2009
   everything in the GPU kernels is in grid-unit. To convert back to length, use
   cfg->unitinmm (scattering/absorption coeff, T, speed etc)
*/

template <int mcxsource>
kernel void mcx_main_loop(uchar media[],float field[],float genergy[],uint n_seed[],
     float4 n_pos[],float4 n_dir[],float4 n_len[],float n_det[], uint detectedphoton[], 
     float srcpattern[],float replayweight[],float photontof[],RandType *seeddata,float *gdebugdata,volatile int *gprogress){

     int idx= blockDim.x * blockIdx.x + threadIdx.x;

     MCXpos  p={0.f,0.f,0.f,-1.f};//{x,y,z}: coordinates in grid unit, w:packet weight
     MCXdir *v=(MCXdir*)(sharedmem+(threadIdx.x<<2));   //{x,y,z}: unitary direction vector in grid unit, nscat:total scat event
     MCXtime f;   //pscat: remaining scattering probability,t: photon elapse time, 
                  //tnext: next accumulation time, ndone: completed photons
     float  energyloss=genergy[idx<<1];
     float  energylaunched=genergy[(idx<<1)+1];

     uint idx1d, idx1dold;   //idx1dold is related to reflection
     uint moves=0;

#ifdef TEST_RACING
     int cc=0;
#endif
     uint  mediaid=gcfg->mediaidorig;
     uint  mediaidold=0,isdet=0;
     float  n1;   //reflection var
     float3 htime;            //time-of-fly for collision test
     float3 rv;               //reciprocal velocity

     //for MT RNG, these will be zero-length arrays and be optimized out
     RandType t[RAND_BUF_LEN];
     RandType photonseed[RAND_BUF_LEN];
     Medium prop;    //can become float2 if no reflection (mua/musp is in 1/grid unit)

     float len, slen;
     float w0,Lmove;
     int   flipdir=-1;
 
     float *ppath=sharedmem+(blockDim.x<<2); // first blockDim.x<<2 stores v for all threads
#ifdef  USE_CACHEBOX
  #ifdef  SAVE_DETECTORS
     float *cachebox=ppath+(gcfg->savedet ? blockDim.x*gcfg->maxmedia: 0);
  #else
     float *cachebox=ppath;
  #endif
     if(gcfg->skipradius2>EPS) clearcache(cachebox,(gcfg->cp1.x-gcfg->cp0.x+1)*(gcfg->cp1.y-gcfg->cp0.y+1)*(gcfg->cp1.z-gcfg->cp0.z+1));
#else
     float accumweight=0.f;
#endif

#ifdef  SAVE_DETECTORS
     ppath+=threadIdx.x*gcfg->maxmedia; // block#2: maxmedia*thread number to store the partial
     if(gcfg->savedet) clearpath(ppath,gcfg->maxmedia);
#endif

     gpu_rng_init(t,n_seed,idx);

     if(launchnewphoton<mcxsource>(&p,v,&f,&rv,&prop,&idx1d,&mediaid,&w0,&Lmove,0,ppath,&energyloss,
       &energylaunched,n_det,detectedphoton,t,photonseed,media,srcpattern,
       idx,(RandType*)n_seed,seeddata,gdebugdata,gprogress)){
         n_seed[idx]=NO_LAUNCH;
	 n_pos[idx]=*((float4*)(&p));
	 n_dir[idx]=*((float4*)(v));
	 n_len[idx]=*((float4*)(&f));
         return;
     }
     rv=float3(__fdividef(1.f,v->x),__fdividef(1.f,v->y),__fdividef(1.f,v->z));
     isdet=mediaid & DET_MASK;
     mediaid &= MED_MASK; // keep isdet to 0 to avoid launching photon ina 

     /*
      using a while-loop to terminate a thread by np.will cause MT RNG to be 3.5x slower
      LL5 RNG will only be slightly slower than for-loop.with photon-move criterion

      we have switched to while-loop since v0.4.9, as LL5 was only minimally effected
      and we do not use MT as the default RNG.
     */

     while(f.ndone<=(gcfg->threadphoton+(idx<gcfg->oddphotons))) {

          GPUDEBUG(("photonid [%d] L=%f w=%e medium=%d\n",(int)f.ndone,f.pscat,p.w,mediaid));

          // dealing with scattering

	  if(f.pscat<=0.f) {  // if this photon has finished his current jump, get next scat length & angles
               if(moves++>gcfg->reseedlimit){
                  moves=0;
                  gpu_rng_reseed(t,n_seed,idx,(p.x+p.y+p.z+p.w)+f.ndone*(v->x+v->y+v->z));
               }
   	       f.pscat=rand_next_scatlen(t); // random scattering probability, unit-less

               GPUDEBUG(("scat L=%f RNG=[%0lX %0lX] \n",f.pscat,t[0],t[1]));
	       if(p.w<1.f){ // if this is not my first jump
                       //random arimuthal angle
	               float cphi,sphi,theta,stheta,ctheta;
                       float tmp0=TWO_PI*rand_next_aangle(t); //next arimuth angle
                       sincosf(tmp0,&sphi,&cphi);
                       GPUDEBUG(("scat phi=%f\n",tmp0));

                       //Henyey-Greenstein Phase Function, "Handbook of Optical 
                       //Biomedical Diagnostics",2002,Chap3,p234, also see Boas2002

                       if(prop.g>EPS){  //if prop.g is too small, the distribution of theta is bad
		           tmp0=(1.f-prop.g*prop.g)/(1.f-prop.g+2.f*prop.g*rand_next_zangle(t));
		           tmp0*=tmp0;
		           tmp0=(1.f+prop.g*prop.g-tmp0)/(2.f*prop.g);

                           // when ran=1, CUDA gives me 1.000002 for tmp0 which produces nan later
                           // detected by Ocelot,thanks to Greg Diamos,see http://bit.ly/cR2NMP
                           tmp0=fmax(-1.f, fmin(1.f, tmp0));

		           theta=acosf(tmp0);
		           stheta=sinf(theta);
		           ctheta=tmp0;
                       }else{
			   theta=acosf(2.f*rand_next_zangle(t)-1.f);
                           sincosf(theta,&stheta,&ctheta);
                       }
                       GPUDEBUG(("scat theta=%f\n",theta));
                       rotatevector(v,stheta,ctheta,sphi,cphi);
                       v->nscat++;
                       rv=float3(__fdividef(1.f,v->x),__fdividef(1.f,v->y),__fdividef(1.f,v->z));
                       if(gcfg->debuglevel & MCX_DEBUG_MOVE)
                           savedebugdata(&p,(uint)f.ndone+idx*gcfg->threadphoton+umin(idx,(idx<gcfg->oddphotons)*idx),gdebugdata);
	       }
	  }

          n1=prop.n;
	  *((float4*)(&prop))=gproperty[mediaid & MED_MASK];
	  
	  len=(gcfg->faststep) ? gcfg->minstep : hitgrid((float3*)&p,(float3*)v,&(htime.x),&rv.x,&flipdir); // propagate the photon to the first intersection to the grid
	  slen=len*prop.mus; //unitless (minstep=grid, mus=1/grid)

          GPUDEBUG(("p=[%f %f %f] -> <%f %f %f>*%f -> hit=[%f %f %f] flip=%d\n",p.x,p.y,p.z,v->x,v->y,v->z,len,htime.x,htime.y,htime.z,flipdir));

          // dealing with absorption
	  slen=fmin(slen,f.pscat);
	  len=slen/prop.mus;
	  *((float3*)(&p)) = (gcfg->faststep || slen==f.pscat) ? float3(p.x+len*v->x,p.y+len*v->y,p.z+len*v->z) : float3(htime.x,htime.y,htime.z);
	  p.w*=expf(-prop.mua*len);
	  f.pscat-=slen;     //remaining probability: sum(s_i*mus_i), unit-less
	  f.t+=len*prop.n*gcfg->oneoverc0; //propagation time  (unit=s)
	  Lmove+=len;

          GPUDEBUG(("update p=[%f %f %f] -> len=%f\n",p.x,p.y,p.z,len));

#ifdef SAVE_DETECTORS
          if(gcfg->savedet)
	      ppath[(mediaid & MED_MASK)-1]+=len; //(unit=grid)
#endif

          mediaidold=mediaid | isdet;
          idx1dold=idx1d;
          idx1d=(int(floorf(p.z))*gcfg->dimlen.y+int(floorf(p.y))*gcfg->dimlen.x+int(floorf(p.x)));
          GPUDEBUG(("idx1d [%d]->[%d]\n",idx1dold,idx1d));
          if(p.x<0||p.y<0||p.z<0||p.x>=gcfg->maxidx.x||p.y>=gcfg->maxidx.y||p.z>=gcfg->maxidx.z){
	      mediaid=0;
	  }else{
	      mediaid=media[idx1d];
	      isdet=mediaid & DET_MASK;
	      mediaid &= MED_MASK;
          }
          GPUDEBUG(("medium [%d]->[%d]\n",mediaidold,mediaid));

          // saving fluence to the voxel when moving out

	  if(idx1d!=idx1dold && idx1dold>0 && mediaidold){
             // if t is within the time window, which spans cfg->maxgate*cfg->tstep.wide
             if(gcfg->save2pt && f.t>=gcfg->twin0 && f.t<gcfg->twin1){
	          float weight;
                  int tshift=(int)(floorf((f.t-gcfg->twin0)*gcfg->Rtstep));
		  if(gcfg->outputtype==otEnergy)
		      weight=w0-p.w;
		  else if(gcfg->seed==SEED_FROM_FILE && gcfg->outputtype==otJacobian){
		      weight=replayweight[(idx*gcfg->threadphoton+min(idx,gcfg->oddphotons-1)+(int)f.ndone)]*Lmove;
                      tshift=(int)(floorf((photontof[(idx*gcfg->threadphoton+min(idx,gcfg->oddphotons-1)+(int)f.ndone)]-gcfg->twin0)*gcfg->Rtstep));
		  }else
		      weight=(prop.mua==0.f) ? 0.f : ((w0-p.w)/(prop.mua));

                  GPUDEBUG(("deposit to [%d] %e, w=%f\n",idx1dold,weight,p.w));

#ifdef TEST_RACING
                  // enable TEST_RACING to determine how many missing accumulations due to race
                  if( (p.x-gcfg->ps.x)*(p.x-gcfg->ps.x)+(p.y-gcfg->ps.y)*(p.y-gcfg->ps.y)+(p.z-gcfg->ps.z)*(p.z-gcfg->ps.z)>gcfg->skipradius2) {
                      field[idx1dold+tshift*gcfg->dimlen.z]+=1.f;
		      cc++;
                  }
#else
  #ifdef USE_ATOMIC
                if(!gcfg->isatomic){
  #endif
                  // set gcfg->skipradius2 to only start depositing energy when dist^2>gcfg->skipradius2 
                  if(gcfg->skipradius2>EPS){
  #ifdef  USE_CACHEBOX
                      if(p.x<gcfg->cp1.x+1.f && p.x>=gcfg->cp0.x &&
		         p.y<gcfg->cp1.y+1.f && p.y>=gcfg->cp0.y &&
			 p.z<gcfg->cp1.z+1.f && p.z>=gcfg->cp0.z){
                         atomicadd(cachebox+(int(p.z-gcfg->cp0.z)*gcfg->cachebox.y
			      +int(p.y-gcfg->cp0.y)*gcfg->cachebox.x+int(p.x-gcfg->cp0.x)),weight);
  #else
                      if((p.x-gcfg->ps.x)*(p.x-gcfg->ps.x)+(p.y-gcfg->ps.y)*(p.y-gcfg->ps.y)+(p.z-gcfg->ps.z)*(p.z-gcfg->ps.z)<=gcfg->skipradius2){
                          accumweight+=p.w*prop.mua; // weight*absorption
  #endif
                      }else{
                          field[idx1dold+tshift*gcfg->dimlen.z]+=weight;
                      }
                  }else{
                      field[idx1dold+tshift*gcfg->dimlen.z]+=weight;
                  }
  #ifdef USE_ATOMIC
               }else{
                  // ifndef CUDA_NO_SM_11_ATOMIC_INTRINSICS
		  atomicadd(& field[idx1dold+tshift*gcfg->dimlen.z], weight);
                  GPUDEBUG(("atomic write to [%d] %e, w=%f\n",idx1dold,weight,p.w));
               }
  #endif
#endif
	     }
	     w0=p.w;
	     Lmove=0.f;
             //f.tnext+=gcfg->minaccumtime*prop.n; // fluence is a temporal-integration, unit=s
	  }
	  
	  // launch new photon when exceed time window or moving from non-zero voxel to zero voxel without reflection

          if((mediaid==0 && (!gcfg->doreflect || (gcfg->doreflect && n1==gproperty[mediaid].w))) || f.t>gcfg->twin1){
              GPUDEBUG(("direct relaunch at idx=[%d] mediaid=[%d], ref=[%d]\n",idx1d,mediaid,gcfg->doreflect));
	      if(launchnewphoton<mcxsource>(&p,v,&f,&rv,&prop,&idx1d,&mediaid,&w0,&Lmove,(mediaidold & DET_MASK),ppath,
	          &energyloss,&energylaunched,n_det,detectedphoton,t,photonseed,media,srcpattern,idx,(RandType*)n_seed,seeddata,gdebugdata,gprogress))
                   break;
              isdet=mediaid & DET_MASK;
              mediaid &= MED_MASK;
	      continue;
	  }

          /*Russian Roulette*/

          if(p.w < gcfg->minenergy){
                if(rand_do_roulette(t)*ROULETTE_SIZE<=1.f)
                   p.w*=ROULETTE_SIZE;
                else{
                   GPUDEBUG(("relaunch after Russian roulette at idx=[%d] mediaid=[%d], ref=[%d]\n",idx1d,mediaid,gcfg->doreflect));
                   if(launchnewphoton<mcxsource>(&p,v,&f,&rv,&prop,&idx1d,&mediaid,&w0,&Lmove,(mediaidold & DET_MASK),ppath,
	                &energyloss,&energylaunched,n_det,detectedphoton,t,photonseed,media,srcpattern,idx,(RandType*)n_seed,seeddata,gdebugdata,gprogress))
                        break;
                   isdet=mediaid & DET_MASK;
                   mediaid &= MED_MASK;
                   continue;
               }
          }

          // do boundary reflection/transmission

	  if(gcfg->doreflect && n1!=gproperty[mediaid].w){
	          float Rtotal=1.f;
	          float cphi,sphi,stheta,ctheta,tmp0,tmp1;

                  *((float4*)(&prop))=gproperty[mediaid]; // optical property across the interface

                  tmp0=n1*n1;
                  tmp1=prop.n*prop.n;
		  cphi=fabs( (flipdir==0) ? v->x : (flipdir==1 ? v->y : v->z)); // cos(si)
		  sphi=1.f-cphi*cphi;            // sin(si)^2

                  len=1.f-tmp0/tmp1*sphi;   //1-[n1/n2*sin(si)]^2 = cos(ti)^2
	          GPUDEBUG(("ref total ref=%f\n",len));

                  if(len>0.f) { // if no total internal reflection
                	ctheta=tmp0*cphi*cphi+tmp1*len;
                	stheta=2.f*n1*prop.n*cphi*sqrtf(len);
                	Rtotal=(ctheta-stheta)/(ctheta+stheta);
       	       		ctheta=tmp1*cphi*cphi+tmp0*len;
       	       		Rtotal=(Rtotal+(ctheta-stheta)/(ctheta+stheta))*0.5f;
	        	GPUDEBUG(("Rtotal=%f\n",Rtotal));
                  } // else, total internal reflection
	          if(Rtotal<1.f && rand_next_reflect(t)>Rtotal){ // do transmission
                        if(mediaid==0){ // transmission to external boundary
                            GPUDEBUG(("transmit to air, relaunch\n"));
		    	    if(launchnewphoton<mcxsource>(&p,v,&f,&rv,&prop,&idx1d,&mediaid,&w0,&Lmove,(mediaidold & DET_MASK),
			        ppath,&energyloss,&energylaunched,n_det,detectedphoton,t,photonseed,
				media,srcpattern,idx,(RandType*)n_seed,seeddata,gdebugdata,gprogress))
                                break;
                            isdet=mediaid & DET_MASK;
                            mediaid &= MED_MASK;
			    continue;
			}
	                GPUDEBUG(("do transmission\n"));
			transmit(v,n1,prop.n,flipdir);
                        rv=float3(__fdividef(1.f,v->x),__fdividef(1.f,v->y),__fdividef(1.f,v->z));
		  }else{ //do reflection
	                GPUDEBUG(("ref faceid=%d p=[%f %f %f] v_old=[%f %f %f]\n",flipdir,p.x,p.y,p.z,v->x,v->y,v->z));
			(flipdir==0) ? (v->x=-v->x) : ((flipdir==1) ? (v->y=-v->y) : (v->z=-v->z)) ;
                        rv=float3(__fdividef(1.f,v->x),__fdividef(1.f,v->y),__fdividef(1.f,v->z));
			(flipdir==0) ?
        		    (p.x=mcx_nextafterf(__float2int_rn(p.x), (v->x > 0.f)-(v->x < 0.f))) :
			    ((flipdir==1) ? 
				(p.y=mcx_nextafterf(__float2int_rn(p.y), (v->x > 0.f)-(v->x < 0.f))) :
				(p.z=mcx_nextafterf(__float2int_rn(p.z), (v->x > 0.f)-(v->x < 0.f))) );
	                GPUDEBUG(("ref p_new=[%f %f %f] v_new=[%f %f %f]\n",p.x,p.y,p.z,v->x,v->y,v->z));
                	idx1d=idx1dold;
		 	mediaid=(media[idx1d] & MED_MASK);
        	  	*((float4*)(&prop))=gproperty[mediaid];
                  	n1=prop.n;
		  }
	  }
     }
     // cachebox saves the total absorbed energy of all time in the sphere r<sradius.
     // in non-atomic mode, cachebox is more accurate than saving to the grid
     // as it is not influenced by race conditions.
     // now I borrow f.tnext to pass this value back
#ifdef  USE_CACHEBOX
     if(gcfg->skipradius2>EPS){
     	f.tnext=0.f;
        savecache(field,cachebox);
     }
#else
     f.tnext=accumweight;
#endif

     genergy[idx<<1]=energyloss;
     genergy[(idx<<1)+1]=energylaunched;

#ifdef TEST_RACING
     n_seed[idx]=cc;
#endif
     n_pos[idx]=*((float4*)(&p));
     n_dir[idx]=*((float4*)(v));
     n_len[idx]=*((float4*)(&f));
}

/**
   assert cuda memory allocation result
*/
void mcx_cu_assess(hipError_t cuerr,const char *file, const int linenum){
     if(cuerr!=hipSuccess){
         mcx_error(-(int)cuerr,(char *)hipGetErrorString(cuerr),file,linenum);
     }
}

/**
  obtain GPU core number per MP, this replaces 
  ConvertSMVer2Cores() in libcudautils to avoid 
  extra dependency.
*/

int mcx_corecount(int v1, int v2){
     int v=v1*10+v2;
     if(v<20)      return 8;
     else if(v<21) return 32;
     else if(v<30) return 48;
     else if(v<50) return 192;
     else          return 128;
}

/**
  obtain GPU core number per MP, this replaces 
  ConvertSMVer2Cores() in libcudautils to avoid 
  extra dependency.
*/

int mcx_smxblock(int v1, int v2){
     int v=v1*10+v2;
     if(v<30)      return 8;
     else if(v<50) return 16;
     else          return 32;
}

/**
  query GPU info and set active GPU
*/
int mcx_list_gpu(Config *cfg, GPUInfo **info){

#if __DEVICE_EMULATION__
    return 1;
#else
    int dev;
    int deviceCount,activedev=0;

    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0){
        MCX_FPRINTF(stderr,"No CUDA-capable GPU device found\n");
        return 0;
    }
    *info=(GPUInfo *)calloc(deviceCount,sizeof(GPUInfo));
    if (cfg->gpuid && cfg->gpuid > deviceCount){
        MCX_FPRINTF(stderr,"Specified GPU ID is out of range\n");
        return 0;
    }
    // scan from the first device
    for (dev = 0; dev<deviceCount; dev++) {
        hipDeviceProp_t dp;
        CUDA_ASSERT(hipGetDeviceProperties(&dp, dev));

	if(cfg->isgpuinfo==3)
	   activedev++;
        else if(cfg->deviceid[dev]=='1'){
           cfg->deviceid[dev]='\0';
           cfg->deviceid[activedev]=dev+1;
           activedev++;
        }
        strncpy((*info)[dev].name,dp.name,MAX_SESSION_LENGTH);
        (*info)[dev].id=dev+1;
	(*info)[dev].devcount=deviceCount;
	(*info)[dev].major=dp.major;
	(*info)[dev].minor=dp.minor;
	(*info)[dev].globalmem=dp.totalGlobalMem;
	(*info)[dev].constmem=dp.totalConstMem;
	(*info)[dev].sharedmem=dp.sharedMemPerBlock;
	(*info)[dev].regcount=dp.regsPerBlock;
	(*info)[dev].clock=dp.clockRate;
	(*info)[dev].sm=dp.multiProcessorCount;
	(*info)[dev].core=dp.multiProcessorCount*mcx_corecount(dp.major,dp.minor);
	(*info)[dev].maxmpthread=dp.maxThreadsPerMultiProcessor;
        (*info)[dev].maxgate=cfg->maxgate;
        (*info)[dev].autoblock=(*info)[dev].maxmpthread / mcx_smxblock(dp.major,dp.minor);
        (*info)[dev].autothread=(*info)[dev].autoblock * mcx_smxblock(dp.major,dp.minor) * (*info)[dev].sm;

        if (strncmp(dp.name, "Device Emulation", 16)) {
	  if(cfg->isgpuinfo){
	    MCX_FPRINTF(stdout,"=============================   GPU Infomation  ================================\n");
	    MCX_FPRINTF(stdout,"Device %d of %d:\t\t%s\n",(*info)[dev].id,(*info)[dev].devcount,(*info)[dev].name);
	    MCX_FPRINTF(stdout,"Compute Capability:\t%u.%u\n",(*info)[dev].major,(*info)[dev].minor);
	    MCX_FPRINTF(stdout,"Global Memory:\t\t%u B\nConstant Memory:\t%u B\n\
Shared Memory:\t\t%u B\nRegisters:\t\t%u\nClock Speed:\t\t%.2f GHz\n",
               (unsigned int)(*info)[dev].globalmem,(unsigned int)(*info)[dev].constmem,
               (unsigned int)(*info)[dev].sharedmem,(unsigned int)(*info)[dev].regcount,(*info)[dev].clock*1e-6f);
	  #if CUDART_VERSION >= 2000
	       MCX_FPRINTF(stdout,"Number of MPs:\t\t%u\nNumber of Cores:\t%u\n",
	          (*info)[dev].sm,(*info)[dev].core);
	  #endif
            MCX_FPRINTF(stdout,"SMX count:\t\t%u\n", (*info)[dev].sm);
	  }
	}
    }
    if(cfg->isgpuinfo==2 && cfg->parentid==mpStandalone){ //list GPU info only
          exit(0);
    }
#ifdef USE_MT_RAND
    if(cfg->nblocksize>N-M){
        mcx_error(-1,"block size can not be larger than 227 when using MT19937 RNG",__FILE__,__LINE__);
    }
#endif

    if(activedev<MAX_DEVICE)
        cfg->deviceid[activedev]='\0';

    return activedev;
#endif
}


/**
   host code for MCX kernels
*/
void mcx_run_simulation(Config *cfg,GPUInfo *gpu){

     int i,iter;
     float  minstep=1.f; //MIN(MIN(cfg->steps.x,cfg->steps.y),cfg->steps.z);
     float4 p0=float4(cfg->srcpos.x,cfg->srcpos.y,cfg->srcpos.z,1.f);
     float4 c0=float4(cfg->srcdir.x,cfg->srcdir.y,cfg->srcdir.z,0.f);
     float3 maxidx=float3(cfg->dim.x,cfg->dim.y,cfg->dim.z);
     float *energy;
     int timegate=0, totalgates, gpuid, gpuphoton=0,threadid=0;

     unsigned int photoncount=0,printnum;
     unsigned int tic,tic0,tic1,toc=0,fieldlen,debuglen=MCX_DEBUG_REC_LEN;
     uint3 cp0=cfg->crop0,cp1=cfg->crop1;
     uint2 cachebox;
     uint3 dimlen;
     float Vvox,fullload=0.f;

     dim3 mcgrid, mcblock;
     dim3 clgrid, clblock;

     int dimxyz=cfg->dim.x*cfg->dim.y*cfg->dim.z;
     
     uchar  *media=(uchar *)(cfg->vol);
     float  *field;
     float4 *Ppos,*Pdir,*Plen,*Plen0;
     uint   *Pseed;
     float  *Pdet;
     RandType *seeddata=NULL;
     uint    detected=0,sharedbuf=0;

     volatile int *progress, *gprogress;

     uchar *gmedia;
     float4 *gPpos,*gPdir,*gPlen;
     uint   *gPseed,*gdetected;
     float  *gPdet,*gsrcpattern,*gfield,*genergy,*greplayw,*greplaytof,*gdebugdata;
     RandType *gseeddata=NULL;
     MCXParam param={cfg->steps,minstep,0,0,cfg->tend,R_C0*cfg->unitinmm,
                     (uint)cfg->issave2pt,(uint)cfg->isreflect,(uint)cfg->isrefint,(uint)cfg->issavedet,1.f/cfg->tstep,
		     p0,c0,maxidx,uint3(0,0,0),cp0,cp1,uint2(0,0),cfg->minenergy,
                     cfg->sradius*cfg->sradius,minstep*R_C0*cfg->unitinmm,cfg->srctype,
		     cfg->srcparam1,cfg->srcparam2,cfg->voidtime,cfg->maxdetphoton,
		     cfg->medianum-1,cfg->detnum,0,0,cfg->reseedlimit,ABS(cfg->sradius+2.f)<EPS /*isatomic*/,
		     (uint)cfg->maxvoidstep,cfg->issaveseed>0,cfg->maxdetphoton*(cfg->medianum+1),cfg->seed,
		     (uint)cfg->outputtype,0,0,cfg->faststep,cfg->debuglevel,(uint)cfg->maxjumpdebug};
     int detreclen=cfg->medianum+1;
     if(param.isatomic)
         param.skipradius2=0.f;

#ifdef _OPENMP
     threadid=omp_get_thread_num();
#endif
     if(threadid<MAX_DEVICE && cfg->deviceid[threadid]=='\0')
           return;

     gpuid=cfg->deviceid[threadid]-1;
     CUDA_ASSERT(hipSetDevice(gpuid));

     if(gpu[gpuid].maxgate==0 && dimxyz>0){
         int needmem=dimxyz+cfg->nthread*sizeof(float4)*4+sizeof(float)*cfg->maxdetphoton*(cfg->medianum+1)+10*1024*1024; /*keep 10M for other things*/
         gpu[gpuid].maxgate=(gpu[gpuid].globalmem-needmem)/(cfg->dim.x*cfg->dim.y*cfg->dim.z);
         gpu[gpuid].maxgate=MIN(((cfg->tend-cfg->tstart)/cfg->tstep+0.5),gpu[gpuid].maxgate);     
     }
     /*only allow the master thread to modify cfg, others are read-only*/
#pragma omp master
{
     if(cfg->exportfield==NULL)
         cfg->exportfield=(float *)calloc(sizeof(float)*cfg->dim.x*cfg->dim.y*cfg->dim.z,gpu[gpuid].maxgate*2);
     if(cfg->exportdetected==NULL)
         cfg->exportdetected=(float*)malloc((cfg->medianum+1)*cfg->maxdetphoton*sizeof(float));
     if(cfg->issaveseed && cfg->seeddata==NULL)
         cfg->seeddata=malloc(cfg->maxdetphoton*sizeof(float)*RAND_BUF_LEN);
     cfg->detectedcount=0;
     cfg->his.detected=0;
     cfg->energytot=0.f;
     cfg->energyabs=0.f;
     cfg->energyesc=0.f;
     cfg->runtime=0;
}
#pragma omp barrier

     if(!cfg->autopilot){
	gpu[gpuid].autothread=cfg->nthread;
	gpu[gpuid].autoblock=cfg->nblocksize;
	gpu[gpuid].maxgate=cfg->maxgate;
     }
     if(gpu[gpuid].autothread%gpu[gpuid].autoblock)
     	gpu[gpuid].autothread=(gpu[gpuid].autothread/gpu[gpuid].autoblock)*gpu[gpuid].autoblock;

     if(cfg->respin>1){
         field=(float *)calloc(sizeof(float)*dimxyz,gpu[gpuid].maxgate*2);
     }else{
         field=(float *)calloc(sizeof(float)*dimxyz,gpu[gpuid].maxgate); //the second half will be used to accumulate
     }

#pragma omp master
{
     fullload=0.f;
     for(i=0;cfg->deviceid[i];i++)
        fullload+=cfg->workload[i];

     if(fullload<EPS){
        for(i=0;cfg->deviceid[i];i++)
            cfg->workload[i]=gpu[cfg->deviceid[i]-1].core;
     }
}
#pragma omp barrier

     fullload=0.f;
     for(i=0;cfg->deviceid[i];i++)
        if(cfg->workload[i]>0.f)
            fullload+=cfg->workload[i];
        else
            mcx_error(-1,"workload was unspecified for an active device",__FILE__,__LINE__);

     gpuphoton=(double)cfg->nphoton*cfg->workload[threadid]/fullload;

     if(gpuphoton==0)
        return;

     param.threadphoton=gpuphoton/gpu[gpuid].autothread/cfg->respin;
     param.oddphotons=gpuphoton/cfg->respin-param.threadphoton*gpu[gpuid].autothread;
     totalgates=(int)((cfg->tend-cfg->tstart)/cfg->tstep+0.5);
#pragma omp master
     if(totalgates>gpu[gpuid].maxgate && cfg->isnormalized){
         MCX_FPRINTF(stderr,"WARNING: GPU memory can not hold all time gates, disabling normalization to allow multiple runs\n");
         cfg->isnormalized=0;
     }
#pragma omp barrier

     fieldlen=dimxyz*gpu[gpuid].maxgate;

     mcgrid.x=gpu[gpuid].autothread/gpu[gpuid].autoblock;
     mcblock.x=gpu[gpuid].autoblock;

     clgrid.x=cfg->dim.x;
     clgrid.y=cfg->dim.y;
     clblock.x=cfg->dim.z;

     if(cfg->debuglevel & MCX_DEBUG_RNG){
#pragma omp master
{
           param.twin0=cfg->tstart;
           param.twin1=cfg->tend;
           Pseed=(uint*)malloc(sizeof(uint)*RAND_SEED_LEN);
           for (i=0; i<RAND_SEED_LEN; i++)
		Pseed[i]=rand();
           CUDA_ASSERT(hipMalloc((void **) &gPseed, sizeof(uint)*RAND_SEED_LEN));
	   CUDA_ASSERT(hipMemcpy(gPseed, Pseed, sizeof(uint)*RAND_SEED_LEN,  hipMemcpyHostToDevice));
           CUDA_ASSERT(hipMalloc((void **) &gfield, sizeof(float)*fieldlen));
           CUDA_ASSERT(hipMemset(gfield,0,sizeof(float)*fieldlen)); // cost about 1 ms
           CUDA_ASSERT(hipMemcpyToSymbol(HIP_SYMBOL(gcfg),   &param, sizeof(MCXParam), 0, hipMemcpyHostToDevice));

           tic=StartTimer();
           MCX_FPRINTF(cfg->flog,"generating %d random numbers ... \t",fieldlen); fflush(cfg->flog);
           mcx_test_rng<<<1,1>>>(gfield,gPseed);
           tic1=GetTimeMillis();
           MCX_FPRINTF(cfg->flog,"kernel complete:  \t%d ms\nretrieving random numbers ... \t",tic1-tic);
           CUDA_ASSERT(hipGetLastError());

           CUDA_ASSERT(hipMemcpy(field, gfield,sizeof(float)*dimxyz*gpu[gpuid].maxgate,hipMemcpyDeviceToHost));
           MCX_FPRINTF(cfg->flog,"transfer complete:\t%d ms\n",GetTimeMillis()-tic);  fflush(cfg->flog);
	   if(cfg->exportfield)
	       memcpy(cfg->exportfield,field,fieldlen*sizeof(float));
	   if(cfg->issave2pt && cfg->parentid==mpStandalone){
               MCX_FPRINTF(cfg->flog,"saving data to file ...\t");
	       mcx_savedata(field,fieldlen,timegate>0,"mc2",cfg);
               MCX_FPRINTF(cfg->flog,"saving data complete : %d ms\n\n",GetTimeMillis()-tic);
               fflush(cfg->flog);
           }
	   CUDA_ASSERT(hipFree(gfield));
	   CUDA_ASSERT(hipFree(gPseed));
	   free(field);
	   free(Pseed);

           CUDA_ASSERT(hipDeviceReset());
}
#pragma omp barrier

	   return;
     }

     Ppos=(float4*)malloc(sizeof(float4)*gpu[gpuid].autothread);
     Pdir=(float4*)malloc(sizeof(float4)*gpu[gpuid].autothread);
     Plen=(float4*)malloc(sizeof(float4)*gpu[gpuid].autothread);
     Plen0=(float4*)malloc(sizeof(float4)*gpu[gpuid].autothread);
     energy=(float*)calloc(gpu[gpuid].autothread<<1,sizeof(float));
     Pdet=(float*)calloc(cfg->maxdetphoton,sizeof(float)*(detreclen));
     Pseed=(uint*)malloc(sizeof(uint)*gpu[gpuid].autothread*RAND_SEED_LEN);

     CUDA_ASSERT(hipMalloc((void **) &gmedia, sizeof(uchar)*(dimxyz)));
     //CUDA_ASSERT(hipBindTexture(0, texmedia, gmedia));
     CUDA_ASSERT(hipMalloc((void **) &gfield, sizeof(float)*fieldlen));
     CUDA_ASSERT(hipMalloc((void **) &gPpos, sizeof(float4)*gpu[gpuid].autothread));
     CUDA_ASSERT(hipMalloc((void **) &gPdir, sizeof(float4)*gpu[gpuid].autothread));
     CUDA_ASSERT(hipMalloc((void **) &gPlen, sizeof(float4)*gpu[gpuid].autothread));
     CUDA_ASSERT(hipMalloc((void **) &gPdet, sizeof(float)*cfg->maxdetphoton*(detreclen)));
     CUDA_ASSERT(hipMalloc((void **) &gdetected, sizeof(uint)));
     CUDA_ASSERT(hipMalloc((void **) &genergy, sizeof(float)*(gpu[gpuid].autothread<<1)));

     CUDA_ASSERT(hipHostAlloc((void **)&progress, sizeof(int), hipHostMallocMapped));
     CUDA_ASSERT(hipHostGetDevicePointer((int **)&gprogress, (int *)progress, 0));
     *progress = 0;

     if(cfg->debuglevel & MCX_DEBUG_MOVE){
         CUDA_ASSERT(hipMalloc((void **) &gdebugdata, sizeof(float)*(debuglen*cfg->maxjumpdebug)));
     }
     if(cfg->issaveseed){
         seeddata=(RandType*)malloc(sizeof(RandType)*cfg->maxdetphoton*RAND_SEED_LEN);
	 CUDA_ASSERT(hipMalloc((void **) &gseeddata, sizeof(RandType)*cfg->maxdetphoton*RAND_SEED_LEN));
     }
     if(cfg->seed==SEED_FROM_FILE){
         CUDA_ASSERT(hipMalloc((void **) &gPseed, sizeof(float)*cfg->nphoton*RAND_SEED_LEN));
	 CUDA_ASSERT(hipMemcpy(gPseed,cfg->replay.seed,sizeof(float)*cfg->nphoton*RAND_SEED_LEN, hipMemcpyHostToDevice));
	 if(cfg->replay.weight){
	     CUDA_ASSERT(hipMalloc((void **) &greplayw, sizeof(float)*cfg->nphoton));
	     CUDA_ASSERT(hipMemcpy(greplayw,cfg->replay.weight,sizeof(float)*cfg->nphoton, hipMemcpyHostToDevice));
	 }
         if(cfg->replay.tof){
	     CUDA_ASSERT(hipMalloc((void **) &greplaytof, sizeof(float)*cfg->nphoton));
	     CUDA_ASSERT(hipMemcpy(greplaytof,cfg->replay.tof,sizeof(float)*cfg->nphoton, hipMemcpyHostToDevice));
	 }
     }else
         CUDA_ASSERT(hipMalloc((void **) &gPseed, sizeof(uint)*gpu[gpuid].autothread*RAND_SEED_LEN));

     if(cfg->srctype==MCX_SRC_PATTERN)
         CUDA_ASSERT(hipMalloc((void **) &gsrcpattern, sizeof(float)*(int)(cfg->srcparam1.w*cfg->srcparam2.w)));

#ifndef SAVE_DETECTORS
#pragma omp master
     if(cfg->issavedet){
           MCX_FPRINTF(stderr,"WARNING: this MCX binary can not save partial path, please use mcx_det or mcx_det_cached\n");
           cfg->issavedet=0;
     }
#pragma omp barrier
#endif

     /*volume is assumbed to be col-major*/
     cachebox.x=(cp1.x-cp0.x+1);
     cachebox.y=(cp1.y-cp0.y+1)*(cp1.x-cp0.x+1);
     dimlen.x=cfg->dim.x;
     dimlen.y=cfg->dim.y*cfg->dim.x;

     dimlen.z=cfg->dim.x*cfg->dim.y*cfg->dim.z;

     param.dimlen=dimlen;
     param.cachebox=cachebox;
     if(p0.x<0.f || p0.y<0.f || p0.z<0.f || p0.x>=cfg->dim.x || p0.y>=cfg->dim.y || p0.z>=cfg->dim.z){
         param.idx1dorig=0;
         param.mediaidorig=0;
     }else{
         param.idx1dorig=(int(floorf(p0.z))*dimlen.y+int(floorf(p0.y))*dimlen.x+int(floorf(p0.x)));
         param.mediaidorig=(cfg->vol[param.idx1dorig] & MED_MASK);
     }

     Vvox=cfg->steps.x*cfg->steps.y*cfg->steps.z; /*Vvox: voxel volume in mm^3*/

     if(cfg->seed>0)
     	srand(cfg->seed+threadid);
     else
        srand(time(0));

     for (i=0; i<gpu[gpuid].autothread; i++) {
	   Ppos[i]=p0;  // initial position
           Pdir[i]=c0;
           Plen[i]=float4(0.f,0.f,param.minaccumtime,0.f);
     }
     tic=StartTimer();
#pragma omp master
{
     mcx_printheader(cfg);

#ifdef MCX_TARGET_NAME
     MCX_FPRINTF(cfg->flog,"- variant name: [%s] compiled for GPU Capability [%d] with CUDA [%d]\n",
         "Fermi",MCX_CUDA_ARCH,CUDART_VERSION);
#else
     MCX_FPRINTF(cfg->flog,"- code name: [Vanilla MCX] compiled for GPU Capacity [%d] with CUDA [%d]\n",
         MCX_CUDA_ARCH,CUDART_VERSION);
#endif
     MCX_FPRINTF(cfg->flog,"- compiled with: RNG [%s] with Seed Length [%d]\n",MCX_RNG_NAME,RAND_SEED_LEN);
#ifdef SAVE_DETECTORS
     MCX_FPRINTF(cfg->flog,"- this version CAN save photons at the detectors\n\n");
#else
     MCX_FPRINTF(cfg->flog,"- this version CAN NOT save photons at the detectors\n\n");
#endif
     fflush(cfg->flog);
}
#pragma omp barrier

     MCX_FPRINTF(cfg->flog,"\nGPU=%d (%s) threadph=%d extra=%d np=%d nthread=%d maxgate=%d repetition=%d\n",gpuid+1,gpu[gpuid].name,param.threadphoton,param.oddphotons,
           gpuphoton,gpu[gpuid].autothread,gpu[gpuid].maxgate,cfg->respin);
     MCX_FPRINTF(cfg->flog,"initializing streams ...\t");
     fflush(cfg->flog);

     CUDA_ASSERT(hipMemcpy(gmedia, media, sizeof(uchar) *dimxyz, hipMemcpyHostToDevice));
     CUDA_ASSERT(hipMemcpy(genergy,energy,sizeof(float) *(gpu[gpuid].autothread<<1), hipMemcpyHostToDevice));
     if(cfg->srcpattern)
         CUDA_ASSERT(hipMemcpy(gsrcpattern,cfg->srcpattern,sizeof(float)*(int)(cfg->srcparam1.w*cfg->srcparam2.w), hipMemcpyHostToDevice));

     CUDA_ASSERT(hipMemcpyToSymbol(HIP_SYMBOL(gproperty), cfg->prop,  cfg->medianum*sizeof(Medium), 0, hipMemcpyHostToDevice));
     CUDA_ASSERT(hipMemcpyToSymbol(HIP_SYMBOL(gdetpos), cfg->detpos,  cfg->detnum*sizeof(float4), 0, hipMemcpyHostToDevice));

     MCX_FPRINTF(cfg->flog,"init complete : %d ms\n",GetTimeMillis()-tic);

     /*
         if one has to simulate a lot of time gates, using the GPU global memory
	 requires extra caution. If the total global memory is bigger than the total
	 memory to save all the snapshots, i.e. size(field)*(tend-tstart)/tstep, one
	 simply sets gpu[gpuid].maxgate to the total gate number; this will run GPU kernel
	 once. If the required memory is bigger than the video memory, set gpu[gpuid].maxgate
	 to a number which fits, and the snapshot will be saved with an increment of 
	 gpu[gpuid].maxgate snapshots. In this case, the later simulations will restart from
	 photon launching and exhibit redundancies.

	 The calculation of the energy conservation will only reflect the last simulation.
     */
     sharedbuf=gpu[gpuid].autoblock*(sizeof(RandType)*RAND_SEED_LEN+sizeof(MCXdir));
#ifdef  USE_CACHEBOX
     if(cfg->sradius>EPS || ABS(cfg->sradius+1.f)<EPS)
        sharedbuf+=sizeof(float)*((cp1.x-cp0.x+1)*(cp1.y-cp0.y+1)*(cp1.z-cp0.z+1));
#endif
     if(cfg->issavedet)
        sharedbuf+=gpu[gpuid].autoblock*sizeof(float)*(cfg->medianum-1);
#ifdef USE_MT_RAND
     sharedbuf+=(N+2)*sizeof(uint); // MT RNG uses N+2 uint in the shared memory
#endif

     MCX_FPRINTF(cfg->flog,"requesting %d bytes of shared memory\n",sharedbuf);

     //simulate for all time-gates in maxgate groups per run
     for(timegate=0;timegate<totalgates;timegate+=gpu[gpuid].maxgate){

       param.twin0=cfg->tstart+cfg->tstep*timegate;
       param.twin1=param.twin0+cfg->tstep*gpu[gpuid].maxgate;
       CUDA_ASSERT(hipMemcpyToSymbol(HIP_SYMBOL(gcfg),   &param,     sizeof(MCXParam), 0, hipMemcpyHostToDevice));

       MCX_FPRINTF(cfg->flog,"lauching MCX simulation for time window [%.2ens %.2ens] ...\n"
           ,param.twin0*1e9,param.twin1*1e9);

       //total number of repetition for the simulations, results will be accumulated to field
       for(iter=0;iter<(int)cfg->respin;iter++){
           CUDA_ASSERT(hipMemset(gfield,0,sizeof(float)*fieldlen)); // cost about 1 ms
           CUDA_ASSERT(hipMemset(gPdet,0,sizeof(float)*cfg->maxdetphoton*(detreclen)));
           if(cfg->issaveseed)
	       CUDA_ASSERT(hipMemset(gseeddata,0,sizeof(RandType)*cfg->maxdetphoton*RAND_BUF_LEN));
           CUDA_ASSERT(hipMemset(gdetected,0,sizeof(float)));
           if(cfg->debuglevel & MCX_DEBUG_MOVE){
	       uint jumpcount=0;
               CUDA_ASSERT(hipMemcpyToSymbol(HIP_SYMBOL(gjumpdebug), &jumpcount, sizeof(uint), 0, hipMemcpyHostToDevice));
           }
 	   CUDA_ASSERT(hipMemcpy(gPpos,  Ppos,  sizeof(float4)*gpu[gpuid].autothread,  hipMemcpyHostToDevice));
	   CUDA_ASSERT(hipMemcpy(gPdir,  Pdir,  sizeof(float4)*gpu[gpuid].autothread,  hipMemcpyHostToDevice));
	   CUDA_ASSERT(hipMemcpy(gPlen,  Plen,  sizeof(float4)*gpu[gpuid].autothread,  hipMemcpyHostToDevice));

           if(cfg->seed!=SEED_FROM_FILE){
             for (i=0; i<gpu[gpuid].autothread*RAND_SEED_LEN; i++)
               Pseed[i]=rand();
	     CUDA_ASSERT(hipMemcpy(gPseed, Pseed, sizeof(uint)*gpu[gpuid].autothread*RAND_SEED_LEN,  hipMemcpyHostToDevice));
           }
           tic0=GetTimeMillis();
           MCX_FPRINTF(cfg->flog,"simulation run#%2d ... \t",iter+1); fflush(cfg->flog);
	   
	   switch(cfg->srctype) {
		case(MCX_SRC_PENCIL): mcx_main_loop<MCX_SRC_PENCIL> <<<mcgrid,mcblock,sharedbuf>>>(gmedia,gfield,genergy,gPseed,gPpos,gPdir,gPlen,gPdet,gdetected,gsrcpattern,greplayw,greplaytof,gseeddata,gdebugdata,gprogress); break;
		case(MCX_SRC_ISOTROPIC): mcx_main_loop<MCX_SRC_ISOTROPIC> <<<mcgrid,mcblock,sharedbuf>>>(gmedia,gfield,genergy,gPseed,gPpos,gPdir,gPlen,gPdet,gdetected,gsrcpattern,greplayw,greplaytof,gseeddata,gdebugdata,gprogress); break;
		case(MCX_SRC_CONE): mcx_main_loop<MCX_SRC_CONE> <<<mcgrid,mcblock,sharedbuf>>>(gmedia,gfield,genergy,gPseed,gPpos,gPdir,gPlen,gPdet,gdetected,gsrcpattern,greplayw,greplaytof,gseeddata,gdebugdata,gprogress); break;
		case(MCX_SRC_GAUSSIAN): mcx_main_loop<MCX_SRC_GAUSSIAN> <<<mcgrid,mcblock,sharedbuf>>>(gmedia,gfield,genergy,gPseed,gPpos,gPdir,gPlen,gPdet,gdetected,gsrcpattern,greplayw,greplaytof,gseeddata,gdebugdata,gprogress); break;
		case(MCX_SRC_PLANAR): mcx_main_loop<MCX_SRC_PLANAR> <<<mcgrid,mcblock,sharedbuf>>>(gmedia,gfield,genergy,gPseed,gPpos,gPdir,gPlen,gPdet,gdetected,gsrcpattern,greplayw,greplaytof,gseeddata,gdebugdata,gprogress); break;
		case(MCX_SRC_PATTERN): mcx_main_loop<MCX_SRC_PATTERN> <<<mcgrid,mcblock,sharedbuf>>>(gmedia,gfield,genergy,gPseed,gPpos,gPdir,gPlen,gPdet,gdetected,gsrcpattern,greplayw,greplaytof,gseeddata,gdebugdata,gprogress); break;
		case(MCX_SRC_FOURIER): mcx_main_loop<MCX_SRC_FOURIER> <<<mcgrid,mcblock,sharedbuf>>>(gmedia,gfield,genergy,gPseed,gPpos,gPdir,gPlen,gPdet,gdetected,gsrcpattern,greplayw,greplaytof,gseeddata,gdebugdata,gprogress); break;
		case(MCX_SRC_ARCSINE): mcx_main_loop<MCX_SRC_ARCSINE> <<<mcgrid,mcblock,sharedbuf>>>(gmedia,gfield,genergy,gPseed,gPpos,gPdir,gPlen,gPdet,gdetected,gsrcpattern,greplayw,greplaytof,gseeddata,gdebugdata,gprogress); break;
		case(MCX_SRC_DISK): mcx_main_loop<MCX_SRC_DISK> <<<mcgrid,mcblock,sharedbuf>>>(gmedia,gfield,genergy,gPseed,gPpos,gPdir,gPlen,gPdet,gdetected,gsrcpattern,greplayw,greplaytof,gseeddata,gdebugdata,gprogress); break;
		case(MCX_SRC_FOURIERX): mcx_main_loop<MCX_SRC_FOURIERX> <<<mcgrid,mcblock,sharedbuf>>>(gmedia,gfield,genergy,gPseed,gPpos,gPdir,gPlen,gPdet,gdetected,gsrcpattern,greplayw,greplaytof,gseeddata,gdebugdata,gprogress); break;
		case(MCX_SRC_FOURIERX2D): mcx_main_loop<MCX_SRC_FOURIERX2D> <<<mcgrid,mcblock,sharedbuf>>>(gmedia,gfield,genergy,gPseed,gPpos,gPdir,gPlen,gPdet,gdetected,gsrcpattern,greplayw,greplaytof,gseeddata,gdebugdata,gprogress); break;
		case(MCX_SRC_ZGAUSSIAN): mcx_main_loop<MCX_SRC_ZGAUSSIAN> <<<mcgrid,mcblock,sharedbuf>>>(gmedia,gfield,genergy,gPseed,gPpos,gPdir,gPlen,gPdet,gdetected,gsrcpattern,greplayw,greplaytof,gseeddata,gdebugdata,gprogress); break;
		case(MCX_SRC_LINE): mcx_main_loop<MCX_SRC_LINE> <<<mcgrid,mcblock,sharedbuf>>>(gmedia,gfield,genergy,gPseed,gPpos,gPdir,gPlen,gPdet,gdetected,gsrcpattern,greplayw,greplaytof,gseeddata,gdebugdata,gprogress); break;
		case(MCX_SRC_SLIT): mcx_main_loop<MCX_SRC_SLIT> <<<mcgrid,mcblock,sharedbuf>>>(gmedia,gfield,genergy,gPseed,gPpos,gPdir,gPlen,gPdet,gdetected,gsrcpattern,greplayw,greplaytof,gseeddata,gdebugdata,gprogress); break;
	   }

#pragma omp master
{
           if((param.debuglevel & MCX_DEBUG_PROGRESS)){
	     int p0 = 0;
	     do{
	       int ndone = *progress;
	       if (ndone > p0){
		  mcx_progressbar(ndone/(param.threadphoton*1.5f),cfg);
		  p0 = ndone;
	       }
               sleep_ms(100);
	     }while (p0 < (param.threadphoton*1.5f));
             mcx_progressbar(1.0f,cfg);
             MCX_FPRINTF(cfg->flog,"\n");
             *progress=0;
           }
}
           CUDA_ASSERT(hipDeviceSynchronize());
	   CUDA_ASSERT(hipMemcpy(&detected, gdetected,sizeof(uint),hipMemcpyDeviceToHost));
           tic1=GetTimeMillis();
	   toc+=tic1-tic0;
           MCX_FPRINTF(cfg->flog,"kernel complete:  \t%d ms\nretrieving fields ... \t",tic1-tic);
           CUDA_ASSERT(hipGetLastError());

           CUDA_ASSERT(hipMemcpy(Plen0,  gPlen,  sizeof(float4)*gpu[gpuid].autothread, hipMemcpyDeviceToHost));
           for(i=0;i<gpu[gpuid].autothread;i++)
	      photoncount+=int(Plen0[i].w+0.5f);

           if(cfg->debuglevel & MCX_DEBUG_MOVE){
               uint debugrec=0;
	       CUDA_ASSERT(hipMemcpyFromSymbol(&debugrec, HIP_SYMBOL(gjumpdebug),sizeof(uint),0,hipMemcpyDeviceToHost));
#pragma omp critical
{
	       if(debugrec>0){
                   debugrec=min(debugrec,cfg->maxjumpdebug);
	           cfg->exportdebugdata=(float*)realloc(cfg->exportdebugdata,(cfg->debugdatalen+debugrec)*debuglen*sizeof(float));
                   CUDA_ASSERT(hipMemcpy(cfg->exportdebugdata+cfg->debugdatalen, gdebugdata,sizeof(float)*debuglen*debugrec,hipMemcpyDeviceToHost));
                   cfg->debugdatalen+=debugrec;
	       }
}
           }
#ifdef SAVE_DETECTORS
           if(cfg->issavedet){
           	CUDA_ASSERT(hipMemcpy(Pdet, gPdet,sizeof(float)*cfg->maxdetphoton*(detreclen),hipMemcpyDeviceToHost));
	        CUDA_ASSERT(hipGetLastError());
		if(cfg->issaveseed)
		    CUDA_ASSERT(hipMemcpy(seeddata, gseeddata,sizeof(RandType)*cfg->maxdetphoton*RAND_BUF_LEN,hipMemcpyDeviceToHost));
		if(detected>cfg->maxdetphoton){
			MCX_FPRINTF(cfg->flog,"WARNING: the detected photon (%d) \
is more than what your have specified (%d), please use the -H option to specify a greater number\t"
                           ,detected,cfg->maxdetphoton);
		}else{
			MCX_FPRINTF(cfg->flog,"detected %d photons, total: %d\t",detected,cfg->detectedcount+detected);
		}
#pragma omp atomic
                cfg->his.detected+=detected;
                detected=MIN(detected,cfg->maxdetphoton);
		if(cfg->exportdetected){
#pragma omp critical
{
                        cfg->exportdetected=(float*)realloc(cfg->exportdetected,(cfg->detectedcount+detected)*detreclen*sizeof(float));
			if(cfg->issaveseed && cfg->seeddata)
			    cfg->seeddata=(RandType*)realloc(cfg->seeddata,(cfg->detectedcount+detected)*sizeof(RandType)*RAND_BUF_LEN);
	                memcpy(cfg->exportdetected+cfg->detectedcount*(detreclen),Pdet,detected*(detreclen)*sizeof(float));
			if(cfg->issaveseed && cfg->seeddata)
			    memcpy(((RandType*)cfg->seeddata)+cfg->detectedcount*RAND_BUF_LEN,seeddata,detected*sizeof(RandType)*RAND_BUF_LEN);
                        cfg->detectedcount+=detected;
}
		}
	   }
#endif

	   //handling the 2pt distributions
           if(cfg->issave2pt){
               CUDA_ASSERT(hipMemcpy(field, gfield,sizeof(float) *dimxyz*gpu[gpuid].maxgate,hipMemcpyDeviceToHost));
               MCX_FPRINTF(cfg->flog,"transfer complete:\t%d ms\n",GetTimeMillis()-tic);  fflush(cfg->flog);

               if(cfg->respin>1){
                   for(i=0;i<(int)fieldlen;i++)  //accumulate field, can be done in the GPU
                      field[fieldlen+i]+=field[i];
               }
           }
       } /*end of respin loop*/

#pragma omp critical
       if(cfg->runtime<toc)
           cfg->runtime=toc;

       if(cfg->respin>1)  //copy the accumulated fields back
           memcpy(field,field+fieldlen,sizeof(float)*fieldlen);

       if(cfg->isnormalized){
           CUDA_ASSERT(hipMemcpy(energy,genergy,sizeof(float)*(gpu[gpuid].autothread<<1),hipMemcpyDeviceToHost));
#pragma omp critical
{
           for(i=0;i<gpu[gpuid].autothread;i++){
               cfg->energyesc+=energy[i<<1];
       	       cfg->energytot+=energy[(i<<1)+1];
           }
	   for(i=0;i<gpu[gpuid].autothread;i++)
               cfg->energyabs+=Plen0[i].z;  // the accumulative absorpted energy near the source
}
       }
       MCX_FPRINTF(cfg->flog,"data normalization complete : %d ms\n",GetTimeMillis()-tic);

       if(cfg->exportfield){
	       for(i=0;i<(int)fieldlen;i++)
#pragma omp atomic
                  cfg->exportfield[i]+=field[i];
       }

       if(param.twin1<cfg->tend){
            CUDA_ASSERT(hipMemset(genergy,0,sizeof(float)*(gpu[gpuid].autothread<<1)));
       }
     } /*end of time-gate group loop*/
#pragma omp barrier

     /*let the master thread to deal with the normalization and file IO*/
#pragma omp master
{
     if(cfg->isnormalized){
	   float scale=1.f;
           MCX_FPRINTF(cfg->flog,"normalizing raw data ...\t");
           cfg->energyabs+=cfg->energytot-cfg->energyesc;
           if(cfg->outputtype==otFlux || cfg->outputtype==otFluence){
               scale=1.f/(cfg->energytot*Vvox*cfg->tstep);
	       if(cfg->unitinmm!=1.f)
		   scale*=cfg->unitinmm; /* Vvox (in mm^3 already) * (Tstep) * (Eabsorp/U) */

               if(cfg->outputtype==otFluence)
		   scale*=cfg->tstep;
	   }else if(cfg->outputtype==otEnergy || cfg->outputtype==otJacobian)
	       scale=1.f/cfg->energytot;

         cfg->normalizer=scale;
	 MCX_FPRINTF(cfg->flog,"normalization factor alpha=%f\n",scale);  fflush(cfg->flog);
         mcx_normalize(cfg->exportfield,scale,fieldlen);
     }
     if(cfg->issave2pt && cfg->parentid==mpStandalone){
         MCX_FPRINTF(cfg->flog,"saving data to file ... %d %d\t",fieldlen,gpu[gpuid].maxgate);
         mcx_savedata(cfg->exportfield,fieldlen,0,"mc2",cfg);
         MCX_FPRINTF(cfg->flog,"saving data complete : %d ms\n\n",GetTimeMillis()-tic);
         fflush(cfg->flog);
     }
     if(cfg->issavedet && cfg->parentid==mpStandalone && cfg->exportdetected){
         cfg->his.unitinmm=cfg->unitinmm;
         cfg->his.savedphoton=cfg->detectedcount;
         if(cfg->issaveseed)
             cfg->his.seedbyte=sizeof(RandType)*RAND_BUF_LEN;

         cfg->his.detected=cfg->detectedcount;
         mcx_savedetphoton(cfg->exportdetected,cfg->seeddata,cfg->detectedcount,0,cfg);
     }
}
#pragma omp barrier

     CUDA_ASSERT(hipMemcpy(Ppos,  gPpos, sizeof(float4)*gpu[gpuid].autothread, hipMemcpyDeviceToHost));
     CUDA_ASSERT(hipMemcpy(Pdir,  gPdir, sizeof(float4)*gpu[gpuid].autothread, hipMemcpyDeviceToHost));
     CUDA_ASSERT(hipMemcpy(Plen,  gPlen, sizeof(float4)*gpu[gpuid].autothread, hipMemcpyDeviceToHost));
     CUDA_ASSERT(hipMemcpy(Pseed, gPseed,sizeof(uint)  *gpu[gpuid].autothread*RAND_SEED_LEN,   hipMemcpyDeviceToHost));
     CUDA_ASSERT(hipMemcpy(energy,genergy,sizeof(float)*(gpu[gpuid].autothread<<1),hipMemcpyDeviceToHost));

#ifdef TEST_RACING
     {
       float totalcount=0.f,hitcount=0.f;
       for (i=0; i<fieldlen; i++)
          hitcount+=field[i];
       for (i=0; i<gpu[gpuid].autothread; i++)
	  totalcount+=Pseed[i];

       MCX_FPRINTF(cfg->flog,"expected total recording number: %f, got %f, missed %f\n",
          totalcount,hitcount,(totalcount-hitcount)/totalcount);
     }
#endif

#pragma omp master
{
     printnum=(gpu[gpuid].autothread<(int)cfg->printnum) ? gpu[gpuid].autothread : cfg->printnum;
     for (i=0; i<(int)printnum; i++) {
            MCX_FPRINTF(cfg->flog,"% 4d[A% f % f % f]C%3d J%5d W% 8f(P%.13f %.13f %.13f)T% 5.3e L% 5.3f %.0f\n", i,
            Pdir[i].x,Pdir[i].y,Pdir[i].z,(int)Plen[i].w,(int)Pdir[i].w,Ppos[i].w, 
            Ppos[i].x,Ppos[i].y,Ppos[i].z,Plen[i].y,Plen[i].x,(float)Pseed[i]);
     }
     // total energy here equals total simulated photons+unfinished photons for all threads
     MCX_FPRINTF(cfg->flog,"simulated %d photons (%d) with %d threads (repeat x%d)\nMCX simulation speed: %.2f photon/ms\n",
             cfg->nphoton,cfg->nphoton,gpu[gpuid].autothread,cfg->respin,(double)cfg->nphoton/cfg->runtime); fflush(cfg->flog);
     MCX_FPRINTF(cfg->flog,"total simulated energy: %.2f\tabsorbed: %5.5f%%\n(loss due to initial specular reflection is excluded in the total)\n",
             cfg->energytot,(cfg->energytot-cfg->energyesc)/cfg->energytot*100.f);fflush(cfg->flog);
     fflush(cfg->flog);
     
     cfg->energyabs=cfg->energytot-cfg->energyesc;
}
#pragma omp barrier

     CUDA_ASSERT(hipFree(gmedia));
     CUDA_ASSERT(hipFree(gfield));
     CUDA_ASSERT(hipFree(gPpos));
     CUDA_ASSERT(hipFree(gPdir));
     CUDA_ASSERT(hipFree(gPlen));
     CUDA_ASSERT(hipFree(gPseed));
     CUDA_ASSERT(hipFree(genergy));
     CUDA_ASSERT(hipFree(gPdet));
     CUDA_ASSERT(hipFree(gdetected));
     if(cfg->issaveseed){
         CUDA_ASSERT(hipFree(gseeddata));
	 free(seeddata);
     }
     if(cfg->seed==SEED_FROM_FILE){
         CUDA_ASSERT(hipFree(greplayw));
         CUDA_ASSERT(hipFree(greplaytof));
     }

     CUDA_ASSERT(hipDeviceReset());

     free(Ppos);
     free(Pdir);
     free(Plen);
     free(Plen0);
     free(Pseed);
     free(Pdet);
     free(energy);
     free(field);
}

