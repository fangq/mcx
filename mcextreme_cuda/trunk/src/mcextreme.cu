#include "hip/hip_runtime.h"
/////////////////////////////////////////////////////////////////////
//
//  MC Extreme  - GPU accelerated Monte-Carlo Simulation
//  
//  Author: Qianqian Fang <fangq at nmr.mgh.harvard.edu>
//  History: 
//    2009/02/14 initial version written in BrookGPU
//    2009/02/15 translated to CUDA
//    2009/02/20 translated to Brook+
//    2009/02/21 added MT random number generator initial version
//    2009/02/24 MT rand now works fine, added FAST_MATH
//    2009/02/25 added CACHE_MEDIA read
//    2009/02/27 early support of boundary reflection
//
// License: unpublished version, use by author's permission only
//
/////////////////////////////////////////////////////////////////////

#include <stdio.h>
#include "br2cu.h"
#include "mt_rand_s.cu"
#include "tictoc.h"

// dimension of the target domain
#define DIMX 128
#define DIMY 128
#define DIMZ 128
/*
#define DIMX 256
#define DIMY 256
#define DIMZ 256
*/

#define DIMYZ (DIMY*DIMZ)
#define DIMXYZ (DIMX*DIMY*DIMZ)
#define INDXYZ(ii,jj,kk)  ((ii)*DIMYZ+(jj)*DIMZ+(kk))
#define MAX_MT_RAND 4294967296
#define R_MAX_MT_RAND 2.32830643653870e-10
#define TWO_PI 6.28318530717959f


#ifdef __DEVICE_EMULATION__
#define MAX_N      1
#define MAX_THREAD 1
#else
#define MAX_N      1024
#define MAX_THREAD 128
#endif
#define MAX_EVENT  1
#define MAX_PROP   256


#ifdef CACHE_MEDIA
#define MAX_MEDIA_CACHE   61440  /*52k for local media read*/
#define MAX_WRITE_CACHE   (MAX_MEDIA_CACHE>>4)
#define MEDIA_BITS  2            /*2^2=4 media types*/
#define MEDIA_PACK  ((8/MEDIA_BITS)>>1)            /*one byte packs 2^MEDIA_PACK voxel*/
#define MEDIA_MOD   ((1<<MEDIA_PACK)-1)    /*one byte packs 2^MEDIA_PACK voxel*/
#define MEDIA_MASK  ((1<<(MEDIA_BITS))-1)
#endif

#define MINUS_SAME_VOXEL -9999.f

#define GPUDIV(a,b)     __fdividef((a),(b))

#ifdef  FAST_MATH      /*define this to use the fast math functions*/
#define	GPULOG(x)       __logf(x)
#define GPUSIN(x)       __sinf(x)
#define	GPUSINCOS(x,a,b)       __sincosf(x,a,b)
#else
#define GPULOG(x)       logf(x)
#define GPUSIN(x)       sinf(x)
#define GPUSINCOS(x,a,b)      sincosf(x,a,b)
#endif


typedef unsigned char uchar;

/******************************
typedef struct PhotonData {
  float4 pos;  // x,y,z,weight
  float4 dir;  // ix,iy,iz,dummy
  float3 len; // resid,tot,count
  uint   seed; // random seed
} Photon;
******************************/

__constant__ float3 gproperty[MAX_PROP];

#ifdef CACHE_MEDIA
__constant__ uchar  gmediacache[MAX_MEDIA_CACHE];
#endif

// pass as many pre-computed values as possible to utilize the constant memory 

kernel void mcx_main_loop(int totalmove,uchar media[],float field[],float3 vsize,float minstep, 
     float lmax, float gg, float gg2,float ggx2, float one_add_gg2, float one_sub_gg2, float one_sub_gg,
     float4 p0,float4 c0,float3 maxidx,uint3 cp0,uint3 cp1,uint2 cachebox,uchar doreflect,
     uint n_seed[],float4 n_pos[],float4 n_dir[],float3 n_len[]){

     int idx= blockDim.x * blockIdx.x + threadIdx.x;

     float4 npos=n_pos[idx];
     float4 ndir=n_dir[idx];
     float3 nlen=n_len[idx];
     float4 npos0;
     float3 htime;

     int i, idx1d, idx1dold,idxorig, mediaid;
     float flipdir,n1,Rtotal;
#ifdef CACHE_MEDIA
     int incache=0,incache0=0,cachebyte=-1,cachebyte0=-1;
#endif
     uint   ran;
     float3 prop;

     float len,cphi,sphi,theta,stheta,ctheta,tmp0,tmp1;


     mt19937si(n_seed[idx]);
     __syncthreads();

     // assuming the initial positions are within the domain
     idx1d=int(floorf(npos.x)*DIMYZ+floorf(npos.y)*DIMZ+floorf(npos.z));
     idxorig=idx1d;
     mediaid=media[idx1d];

#ifdef CACHE_MEDIA
     if(npos.x>=cp0.x && npos.x<=cp1.x && npos.y>=cp0.y && npos.y<=cp1.y && npos.z>=cp0.z && npos.z<=cp1.z){
	  incache=1;
          incache0=1;
          cachebyte=int(floorf(npos.x-cp0.x)*cachebox.y+floorf(npos.y-cp0.y)*cachebox.x+floorf(npos.z-cp0.z));
          cachebyte0=cachebyte;
          mediaid=(int)gmediacache[cachebyte>>MEDIA_PACK];
          mediaid=(mediaid >> (cachebyte & MEDIA_MOD)*MEDIA_BITS) & MEDIA_MASK;
     }
#endif

     if(mediaid==0) {
          return; /* the initial position is not within the medium*/
     }

     for(i=0;i<totalmove;i++){
	  if(nlen.x<=0.f) {  /* if this photon finished the current jump */

	       ran=mt19937s(); /*random number [0,MAX_MT_RAND)*/

   	       nlen.x=-GPULOG(ran*R_MAX_MT_RAND); /*probability of the next jump*/

	       if(npos.w<1.f){ /*weight*/
                       /*random arimuthal angle*/
                       ran=mt19937s();
		       tmp0=TWO_PI*ran*R_MAX_MT_RAND; /*will be reused to minimize register*/
                       GPUSINCOS(tmp0,&sphi,&cphi);

                       /*Henyey-Greenstein Phase Function, "Handbook of Optical Biomedical Diagnostics",2002,Chap3,p234*/
                       /*see Boas2003*/
		       ran=mt19937s();
                       if(gg>1e-10){
		           tmp0=GPUDIV(one_sub_gg2,(one_sub_gg+ggx2*ran*R_MAX_MT_RAND));
		           tmp0*=tmp0;
		           tmp0=GPUDIV((one_add_gg2-tmp0),ggx2);
		           theta=acosf(tmp0);
		           stheta=GPUSIN(theta);
		           ctheta=tmp0;
                       }else{
			   theta=TWO_PI*ran*R_MAX_MT_RAND;
                           GPUSINCOS(theta,&stheta,&ctheta);
                       }
		       if( ndir.z>-1.f && ndir.z<1.f ) {
		           tmp0=1.f-ndir.z*ndir.z;   /*reuse tmp to minimize registers*/
		           tmp1=rsqrtf(tmp0);
		           tmp1=stheta*tmp1;
			   if(stheta>1e-20) {  /*strange: if stheta=0, I will get nan :(  FQ */
			     ndir=float4(
				tmp1*(ndir.x*ndir.z*cphi - ndir.y*sphi) + ndir.x*ctheta,
				tmp1*(ndir.y*ndir.z*cphi + ndir.x*sphi) + ndir.y*ctheta,
				-tmp1*tmp0*cphi                         + ndir.z*ctheta,
				ndir.w
				);
                             }
		       }else{
			   ndir=float4(stheta*cphi,stheta*sphi,ctheta,ndir.w);
 		       }
                       ndir.w++;
	       }
	  }


          n1=prop.z;
	  prop=gproperty[mediaid];
	  len=minstep*prop.y;

          npos0=npos;
	  if(len>nlen.x){  /*scattering ends in this voxel*/
               tmp0=GPUDIV(nlen.x,prop.y);
   	       npos=float4(npos.x+ndir.x*tmp0,npos.y+ndir.y*tmp0,npos.z+ndir.z*tmp0,npos.w*expf(-prop.x * tmp0 ));
	       nlen.x=MINUS_SAME_VOXEL;
	       nlen.y+=tmp0;
	  }else{                      /*otherwise, move minstep*/
   	       npos=float4(npos.x+ndir.x,npos.y+ndir.y,npos.z+ndir.z,npos.w*expf(-prop.x * minstep ));
	       nlen.x-=len;     /*remaining probability*/
	       nlen.y+=minstep; /*total moved length along the current jump*/
               idx1dold=idx1d;
               idx1d=int(floorf(npos.x)*DIMYZ+floorf(npos.y)*DIMZ+floorf(npos.z));
#ifdef CACHE_MEDIA     
               if(npos.x>=cp0.x && npos.x<=cp1.x && npos.y>=cp0.y && npos.y<=cp1.y && npos.z>=cp0.z && npos.z<=cp1.z){
                    incache=1;
                    cachebyte=int(floorf(npos.x-cp0.x)*cachebox.y+floorf(npos.y-cp0.y)*cachebox.x+floorf(npos.z-cp0.z));
               }else{
		    incache=0;
               }
#endif
	  }

#ifdef CACHE_MEDIA
          if(incache){
		mediaid=(int)gmediacache[cachebyte>>MEDIA_PACK];
                mediaid=(mediaid >> (cachebyte & MEDIA_MOD)*MEDIA_BITS) & MEDIA_MASK;
          }else{
#endif
                mediaid=media[idx1d];
#ifdef CACHE_MEDIA
          }
#endif

	  if(mediaid==0||nlen.y>lmax||npos.x<0||npos.y<0||npos.z<0||npos.x>maxidx.x||npos.y>maxidx.y||npos.z>maxidx.z){
	      /*if hit the boundary or exit the domain, launch a new one*/

              /*time to hit the wall in each direction*/
              htime.x=(ndir.x>1e-10||ndir.x<-1e-10)?(floorf(npos.x)+(ndir.x>0.f)-npos.x)/ndir.x:1e10; /*this approximates*/
              htime.y=(ndir.y>1e-10||ndir.y<-1e-10)?(floorf(npos.y)+(ndir.y>0.f)-npos.y)/ndir.y:1e10;
              htime.z=(ndir.z>1e-10||ndir.z<-1e-10)?(floorf(npos.z)+(ndir.z>0.f)-npos.z)/ndir.z:1e10;
              tmp0=fminf(fminf(htime.x,htime.y),htime.z);
              flipdir=(tmp0==htime.x?1.f:(tmp0==htime.y?2.f:(tmp0==htime.z&&idx1d!=idx1dold)?3.f:0.f));
              prop=gproperty[mediaid];

#ifdef __DEVICE_EMULATION__
              printf("--> ID%d J%d C%d len %f flip %f %f!=%f dir=%f %f %f \n",idx,(int)ndir.w,
                  (int)nlen.z,nlen.y, flipdir, n1,prop.z,ndir.x,ndir.y,ndir.z);
#endif

              /*I don't have the luxury to declare more vars in a kernel, so, I recycled some of old ones*/

              if(doreflect&&nlen.y<lmax && flipdir>0.f && n1!=prop.z){
                  tmp0=n1*n1;
                  tmp1=prop.z*prop.z;
                  if(flipdir>=3.f) { /*flip in z axis*/
                     cphi=fabs(ndir.z);
                     sphi=ndir.x*ndir.x+ndir.y*ndir.y;
                     ndir.z=-ndir.z;
                  }else if(flipdir>=2.f){ /*flip in y axis*/
                     cphi=fabs(ndir.y);
       	       	     sphi=ndir.x*ndir.x+ndir.z*ndir.z;
                     ndir.y=-ndir.y;
                  }else if(flipdir>=1.f){ /*flip in x axis*/
                     cphi=fabs(ndir.x);               /*cos(si)*/
                     sphi=ndir.y*ndir.y+ndir.z*ndir.z; /*sin(si)^2*/
                     ndir.x=-ndir.x;
                  }
                  npos=npos0;   /*move back*/
                  idx1d=idx1dold;
                  len=1.f-GPUDIV(tmp0,tmp1)*sphi;   /*1-[n1/n2*sin(si)]^2*/
                  if(len>0.f) {
                     ctheta=tmp0*cphi*cphi+tmp1*len;
                     stheta=2.f*n1*prop.z*cphi*sqrtf(len);
                     Rtotal=GPUDIV(ctheta-stheta,ctheta+stheta);
       	       	     ctheta=tmp1*cphi*cphi+tmp0*len;
       	       	     Rtotal=(Rtotal+GPUDIV(ctheta-stheta,ctheta+stheta))/2.f;
#ifdef __DEVICE_EMULATION__
printf("  dir=%f %f %f htime=%f %f %f Rs=%f\n",ndir.x,ndir.y,ndir.z,htime.x,htime.y,htime.z,Rtotal);
printf("  ID%d J%d C%d flip=%3f (%d %d) cphi=%f sphi=%f npos=%f %f %f npos0=%f %f %f\n",idx,(int)ndir.w,(int)nlen.z,
            flipdir,idx1dold,idx1d,cphi,sphi,npos.x,npos.y,npos.z,npos0.x,npos0.y,npos0.z);
#endif
                     npos.w*=Rtotal;
                  } /* else, total internal reflection, no loss*/
                  mediaid=media[idx1d];
                  prop=gproperty[mediaid];
                  n1=prop.z;
              }else{
	          npos=p0;
	          ndir=c0;
	          nlen=float3(0.f,0.f,nlen.z+1);
                  idx1d=idxorig;
#ifdef CACHE_MEDIA
	          cachebyte=cachebyte0;
	          incache=incache0;
#endif
              }
	  }else if(nlen.x>0){
              field[idx1d]+=npos.w;
	  }
     }
     n_seed[idx]=(ran&0xffffffffu);
     n_pos[idx]=npos;
     n_dir[idx]=ndir;
     n_len[idx]=nlen;
}

void savedata(float *dat,int len,char *name){
     FILE *fp;
     fp=fopen(name,"wb");
     fwrite(dat,sizeof(float),len,fp);
     fclose(fp);
}

int main (int argc, char *argv[]) {

     float3 vsize=float3(1.f,1.f,1.f);
     float  minstep=1.f;
     float  lmax=1000.f;
     float  gg=0.98f;
     float4 p0=float4(DIMX/2,DIMY/2,DIMZ/4,1.f);
     float4 c0=float4(0.f,0.f,1.f,0.f);
     float3 maxidx=float3(DIMX-1,DIMY-1,DIMZ-1);
     float3 property[MAX_PROP]={float3(0.f,0.f,1.0f),float3(0.009f,0.75f,1.37f),  // the 1st is air
                                float3(0.006f,0.75f,1.37f),float3(0.009f,0.95f,1.37f)};

     int i,j,k;
     int total=MAX_EVENT;
     int photoncount=0;
     int tic;
//     uint3 cp0=uint3(DIMX/2-30,DIMY/2-30,DIMZ/4),cp1=uint3(DIMX/2+30,DIMY/2+30,DIMZ/4+60);
     uint3 cp0=uint3(DIMX/2-10,DIMY/2-10,DIMZ/4),cp1=uint3(DIMX/2+10,DIMY/2+10,DIMZ/4+20);
     uint2 cachebox;

     dim3 GridDim(MAX_N/MAX_THREAD);
     dim3 BlockDim(MAX_THREAD);

     uchar  media[DIMXYZ];
     float  field[DIMXYZ];
#ifdef CACHE_MEDIA
     int count;
     uchar  mediacache[MAX_MEDIA_CACHE];
#endif

     float4 Ppos[MAX_N];
     float4 Pdir[MAX_N];
     float3 Plen[MAX_N];
     uint   Pseed[MAX_N];

     if(argc>1){
	   total=atoi(argv[1]);
     }

#ifdef CACHE_MEDIA
     printf("requested constant memory cache: %d (max allowed %d)\n",
         (cp1.x-cp0.x+1)*(cp1.y-cp0.y+1)*(cp1.z-cp0.z+1),(MAX_MEDIA_CACHE<<MEDIA_PACK));
     if((cp1.x-cp0.x+1)*(cp1.y-cp0.y+1)*(cp1.z-cp0.z+1)> (MAX_MEDIA_CACHE<<MEDIA_PACK)){
	printf("the requested cache size is too big\n");
	exit(1);
     }
#endif

     uchar *gmedia;
     hipMalloc((void **) &gmedia, sizeof(uchar)*(DIMXYZ));
     float *gfield;
     hipMalloc((void **) &gfield, sizeof(float)*(DIMXYZ));

     float4 *gPpos;
     hipMalloc((void **) &gPpos, sizeof(float4)*(MAX_N));
     float4 *gPdir;
     hipMalloc((void **) &gPdir, sizeof(float4)*(MAX_N));
     float3 *gPlen;
     hipMalloc((void **) &gPlen, sizeof(float3)*(MAX_N));
     uint   *gPseed;
     hipMalloc((void **) &gPseed, sizeof(uint)*(MAX_N));


     memset(field,0,sizeof(float)*DIMXYZ);
     memset(media,0,sizeof(uchar)*DIMXYZ);

     for (i=DIMX/4; i<3*DIMX/4; i++)
      for (j=DIMY/4; j<3*DIMY/4; j++)
       for (k=DIMZ/4; k<3*DIMZ/4; k++) {
           media[INDXYZ(i,j,k)]=1; 
       }

     cachebox.x=(cp1.z-cp0.z+1);
     cachebox.y=(cp1.y-cp0.y+1)*(cp1.z-cp0.z+1);

#ifdef CACHE_MEDIA
     count=0;
     memset(mediacache,0,MAX_MEDIA_CACHE);
     for (i=cp0.x; i<=cp1.x; i++)
      for (j=cp0.y; j<=cp1.y; j++)
       for (k=cp0.z; k<=cp1.z; k++) {
//         printf("[%d %d %d]: %d %d %d %d (%d)\n",i,j,k,count,MEDIA_MASK,count>>MEDIA_PACK,(count & MEDIA_MOD)*MEDIA_BITS,
//                (media[INDXYZ(i,j,k)] & MEDIA_MASK )<<((count & MEDIA_MOD)*MEDIA_BITS) );
           mediacache[count>>MEDIA_PACK] |=  (media[INDXYZ(i,j,k)] & MEDIA_MASK )<<((count & MEDIA_MOD)*MEDIA_BITS );
           count++;
       }
#endif

     srand(time(0));
     for (i=0; i<MAX_N; i++) {
	   Ppos[i]=p0;  /* initial position */
           Pdir[i]=c0;
           Plen[i]=float3(0.f,0.f,0.f);
	   Pseed[i]=rand();
     }

     tic=GetTimeMillis();

     hipMemcpy(gPpos,  Ppos,  sizeof(float4)*MAX_N,  hipMemcpyHostToDevice);
     hipMemcpy(gPdir,  Pdir,  sizeof(float4)*MAX_N,  hipMemcpyHostToDevice);
     hipMemcpy(gPlen,  Plen,  sizeof(float3)*MAX_N,  hipMemcpyHostToDevice);
     hipMemcpy(gPseed, Pseed, sizeof(uint)*MAX_N,     hipMemcpyHostToDevice);
     hipMemcpy(gfield, field, sizeof(float)*DIMXYZ, hipMemcpyHostToDevice);
     hipMemcpy(gmedia, media, sizeof(uchar)*DIMXYZ,hipMemcpyHostToDevice);
     hipMemcpyToSymbol(HIP_SYMBOL(gproperty), property, MAX_PROP*sizeof(float3), 0, hipMemcpyHostToDevice);
#ifdef CACHE_MEDIA
     hipMemcpyToSymbol(HIP_SYMBOL(gmediacache), mediacache, MAX_MEDIA_CACHE, 0, hipMemcpyHostToDevice);
#endif

     printf("complete hipMemcpy : %d ms\n",GetTimeMillis()-tic);

     mcx_main_loop<<<GridDim,BlockDim>>>(total,gmedia,gfield,vsize,minstep,lmax,gg,gg*gg,2.f*gg,\
        	 1.f+gg*gg,1.f-gg*gg,1.f-gg,p0,c0,maxidx,cp0,cp1,cachebox,0,gPseed,gPpos,gPdir,gPlen);

     printf("complete launching kernels : %d ms\n",GetTimeMillis()-tic);

     hipMemcpy(Ppos,  gPpos, sizeof(float4)*MAX_N, hipMemcpyDeviceToHost);

     printf("complete retrieving pos : %d ms\n",GetTimeMillis()-tic);

     hipMemcpy(Pdir,  gPdir, sizeof(float4)*MAX_N, hipMemcpyDeviceToHost);
     hipMemcpy(Plen,  gPlen, sizeof(float3)*MAX_N, hipMemcpyDeviceToHost);
     hipMemcpy(Pseed, gPseed,sizeof(uint)*MAX_N,   hipMemcpyDeviceToHost);
     hipMemcpy(field, gfield,sizeof(float)*DIMXYZ,hipMemcpyDeviceToHost);

     printf("complete retrieving all : %d ms\n",GetTimeMillis()-tic);

     for (i=0; i<MAX_N; i++) {
	  photoncount+=(int)Plen[i].z;
     }
     total=MAX_N<16?MAX_N:16;
     for (i=0; i<total; i++) {
           printf("% 4d[A% f % f % f]C%3d J%3d% 8f(P% 6.3f % 6.3f % 6.3f)T% 5.3f L% 5.3f %f %f\n", i,
            Pdir[i].x,Pdir[i].y,Pdir[i].z,(int)Plen[i].z,(int)Pdir[i].w,Ppos[i].w, 
            Ppos[i].x,Ppos[i].y,Ppos[i].z,Plen[i].y,Plen[i].x,(float)Pseed[i], Pdir[i].x*Pdir[i].x+Pdir[i].y*Pdir[i].y+Pdir[i].z*Pdir[i].z);
     }
     printf("simulating total photon %d\n",photoncount);
     savedata(field,DIMX*DIMY*DIMZ,"field.dat");

     hipFree(gmedia);
#ifdef CACHE_MEDIA
     hipFree(gmediacache);
#endif
     hipFree(gfield);
     hipFree(gPpos);
     hipFree(gPdir);
     hipFree(gPlen);
     hipFree(gPseed);

     return 0;
}
